#ifdef _WIN32
#define PLUGIN_API extern "C" __declspec(dllexport)
#elif __linux__
#define PLUGIN_API extern "C"
#endif

#include "common.cuh"
#include "control.cuh"
#include "collective_variable/collective_variable.cuh"
#include "MD_core/MD_core.cuh"
#include "neighbor_list/neighbor_list.cuh"
#include <sstream>

#include "Python.h"

static MD_INFORMATION* md_info = NULL;
static CONTROLLER* controller = NULL;
static COLLECTIVE_VARIABLE_CONTROLLER* cv_controller = NULL;
static NEIGHBOR_LIST* neighbor_list = NULL;
static int is_initialized = 0;


struct SpongeCVObject
{
    PyObject_HEAD
    COLLECTIVE_VARIABLE_PROTOTYPE* cv;
};

static PyObject *SpongeCVNew(PyTypeObject *type, PyObject *args, PyObject *kwds)
{
    SpongeCVObject *self;
    self = (SpongeCVObject *) type->tp_alloc(type, 0);
    if (self != NULL)
    {
        self->cv = NULL;
    }
    return (PyObject *) self;
}

static int SpongeCVInit(SpongeCVObject *self, PyObject *args, PyObject *kwds)
{
    static char *kwlist[] = {(char*)"ptr", NULL};
    intptr_t tmp;
    if (!PyArg_ParseTupleAndKeywords(args, kwds, "L", kwlist, &tmp))
        return -1;

    self->cv = (COLLECTIVE_VARIABLE_PROTOTYPE*) tmp;

    return 0;
}

static PyObject* SpongeCVGetValue(SpongeCVObject *self, PyObject *Py_UNUSED(ignored))
{
    return Py_BuildValue("f", self->cv->value);
}

static PyObject* SpongeCVSetValue(SpongeCVObject *self, PyObject *args, PyObject *kwds)
{
    static char *kwlist[] = {(char*)"value", NULL};
    float value;
    if (!PyArg_ParseTupleAndKeywords(args, kwds, "f", kwlist, &value))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    self->cv->value = value;
    
    return Py_BuildValue("");
}

static PyObject* SpongeCVGetLastUpdateStep(SpongeCVObject *self, PyObject *args, PyObject *kwds)
{
    static char *kwlist[] = {(char*)"need", NULL};
    int need;
    if (!PyArg_ParseTupleAndKeywords(args, kwds, "i", kwlist, &need))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    return Py_BuildValue("i", self->cv->last_update_step[(COLLECTIVE_VARIABLE_NEED)need]);
}

static PyObject* SpongeCVSetLastUpdateStep(SpongeCVObject *self, PyObject *args, PyObject *kwds)
{
    static char *kwlist[] = {(char*)"need", (char*)"step", NULL};
    int need;
    int step;
    if (!PyArg_ParseTupleAndKeywords(args, kwds, "ii", kwlist, &need, &step))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    for (int i = 1; i <= 8; i <<= 1)
    {
        if (need && i)
            self->cv->last_update_step[(COLLECTIVE_VARIABLE_NEED)i] = step;
    }
    return Py_BuildValue("");
}

static PyObject* SpongeCVGetStream(SpongeCVObject *self, PyObject *Py_UNUSED(ignored))
{
    return Py_BuildValue("L", (intptr_t)self->cv->hip_stream);
}

static PyObject* SpongeCVDeviceValue(SpongeCVObject *self, PyObject *Py_UNUSED(ignored))
{
    return Py_BuildValue("((Lizi),i,s)",(intptr_t)self->cv->d_value, sizeof(float), NULL, controller->working_device, 1, "f");
}

static PyObject* SpongeCVCrdGrads(SpongeCVObject *self, PyObject *Py_UNUSED(ignored))
{
    return Py_BuildValue("((Lizi),(ii),s)",(intptr_t)self->cv->crd_grads, sizeof(float) * 3 * md_info->atom_numbers, NULL, controller->working_device, md_info->atom_numbers, 3, "f");
}

static PyObject* SpongeCVBoxGrads(SpongeCVObject *self, PyObject *Py_UNUSED(ignored))
{
    return Py_BuildValue("((Lizi),i,s)",(intptr_t)self->cv->box_grads, sizeof(float) * 3, NULL, controller->working_device, 3, "f");
}

static PyMethodDef SpongeCVMethods[] = {
    {"_get_value", (PyCFunction)SpongeCVGetValue, METH_NOARGS, ""},
    {"_set_value", (PyCFunction)SpongeCVSetValue, METH_VARARGS|METH_KEYWORDS, ""},
    {"_get_last_update_step", (PyCFunction)SpongeCVGetLastUpdateStep, METH_VARARGS|METH_KEYWORDS, ""},
    {"_set_last_update_step", (PyCFunction)SpongeCVSetLastUpdateStep, METH_VARARGS|METH_KEYWORDS, ""},
    {"_get_cuda_stream", (PyCFunction)SpongeCVGetStream, METH_NOARGS, ""},
    {"_d_value", (PyCFunction)SpongeCVDeviceValue, METH_NOARGS, ""},
    {"_crd_grads", (PyCFunction)SpongeCVCrdGrads, METH_NOARGS, ""},
    {"_box_grads", (PyCFunction)SpongeCVBoxGrads, METH_NOARGS, ""},
    {NULL,NULL,0,NULL}
};

static PyTypeObject SpongeCVType = {
    PyVarObject_HEAD_INIT(NULL, 0)
    "Sponge._COLLECTIVE_VARIABLE_PROTOTYPE",
    sizeof(SpongeCVObject),0,0,0,
    0,0,0,0,0,0,0,0,0,0,
    0,0,0,0,0,0,0,0,0,0,
    0,SpongeCVMethods,0,0,0,0,0,0,0,(initproc)SpongeCVInit,
    0,SpongeCVNew,0,0,0,
};

static PyObject* Atom_Numbers(PyObject* self, PyObject*args)
{
    return Py_BuildValue("i", md_info->atom_numbers);
}

static PyObject* Steps(PyObject* self, PyObject*args)
{
    return Py_BuildValue("i", md_info->sys.steps);
}

static PyObject* Box_Length(PyObject* self, PyObject*args)
{
    return Py_BuildValue("(fff)", md_info->sys.box_length.x, md_info->sys.box_length.y, md_info->sys.box_length.z);
}

static PyObject* Neighbor_List_Atom_Numbers(PyObject* self, PyObject*args)
{
    return Py_BuildValue("((Lizi),(ii),s)",(intptr_t)neighbor_list->d_nl, sizeof(ATOM_GROUP) * md_info->atom_numbers, NULL,
        controller->working_device, md_info->atom_numbers, sizeof(ATOM_GROUP) / sizeof(int), "i");
}

static PyObject* Neighbor_List_Atom_Serials(PyObject* self, PyObject*args)
{
    return Py_BuildValue("((Lizi),(ii),s)",(intptr_t)neighbor_list->h_nl->atom_serial, sizeof(int) * md_info->atom_numbers * neighbor_list->max_neighbor_numbers, NULL,
        controller->working_device, md_info->atom_numbers, neighbor_list->max_neighbor_numbers, "i");
}

static PyObject* Coordinate(PyObject* self, PyObject*args)
{
    return Py_BuildValue("((Lizi),(ii),s)",(intptr_t)md_info->crd, sizeof(float) * 3 * md_info->atom_numbers, NULL, controller->working_device, md_info->atom_numbers, 3, "f");
}

static PyObject* Velocity(PyObject* self, PyObject*args)
{
    return Py_BuildValue("((Lizi),(ii),s)",(intptr_t)md_info->vel, sizeof(float) * 3 * md_info->atom_numbers, NULL, controller->working_device, md_info->atom_numbers, 3, "f");
}

static PyObject* Acceleration(PyObject* self, PyObject*args)
{
    return Py_BuildValue("((Lizi),(ii),s)",(intptr_t)md_info->acc, sizeof(float) * 3 * md_info->atom_numbers, NULL, controller->working_device, md_info->atom_numbers, 3, "f");
}

static PyObject* Force(PyObject* self, PyObject*args)
{
    return Py_BuildValue("((Lizi),(ii),s)",(intptr_t)md_info->frc, sizeof(float) * 3 * md_info->atom_numbers, NULL, controller->working_device, md_info->atom_numbers, 3, "f");
}

static PyObject* Mass(PyObject* self, PyObject*args)
{
    return Py_BuildValue("((Lizi),i,s)",(intptr_t)md_info->d_mass, sizeof(float) * md_info->atom_numbers, NULL, controller->working_device, md_info->atom_numbers, "f");
}

static PyObject* Charge(PyObject* self, PyObject*args)
{
    return Py_BuildValue("((Lizi),i,s)",(intptr_t)md_info->d_charge, sizeof(float) * md_info->atom_numbers, NULL, controller->working_device, md_info->atom_numbers, "f");
}

static PyObject* Atom_Energy(PyObject* self, PyObject*args)
{
    return Py_BuildValue("((Lizi),i,s)",(intptr_t)md_info->d_atom_energy, sizeof(float) * md_info->atom_numbers, NULL, controller->working_device, md_info->atom_numbers, "f");
}

static PyObject* Atom_Virial(PyObject* self, PyObject*args)
{
    return Py_BuildValue("((Lizi),i,s)",(intptr_t)md_info->d_atom_virial, sizeof(float) * md_info->atom_numbers, NULL, controller->working_device, md_info->atom_numbers, "f");
}

static PyObject* Working_Device(PyObject* self, PyObject*args)
{
    return Py_BuildValue("i", controller->working_device);
}

static PyObject* Control_Printf(PyObject* self, PyObject* args, PyObject* kw)
{
    static char* kwlist[] = {(char*)"toprint", NULL};
    char *buffer;
    if (!PyArg_ParseTupleAndKeywords(args, kw, "s",kwlist, &buffer))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    controller->printf("%s", buffer);
    return Py_BuildValue("");
}

static PyObject* Control_Warn(PyObject* self, PyObject* args, PyObject* kw)
{
    static char* kwlist[] = {(char*)"towarn", NULL};
    char *buffer;
    if (!PyArg_ParseTupleAndKeywords(args, kw, "s",kwlist, &buffer))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    controller->Warn(buffer);
    return Py_BuildValue("");
}

static PyObject* Control_Error(PyObject* self, PyObject* args, PyObject* kw)
{
    static char* kwlist[] = {(char*)"error", (char*)"error_by", (char*)"error_hint", NULL};
    int error_num;
    char *buffer1;
    char *buffer2;
    if (!PyArg_ParseTupleAndKeywords(args, kw, "iss",kwlist, &error_num, &buffer1, &buffer2))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    controller->Throw_SPONGE_Error(error_num, buffer1, buffer2);
    return Py_BuildValue("");
}

static PyObject* Command_Exist(PyObject* self, PyObject* args, PyObject* kw)
{
    static char* kwlist[] = {(char*)"command", NULL};
    char *buffer;
    if (!PyArg_ParseTupleAndKeywords(args, kw, "s", kwlist, &buffer))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    if (controller->Command_Exist(buffer))
        Py_RETURN_TRUE;
    else
        Py_RETURN_FALSE;
}

static PyObject* Command(PyObject* self, PyObject* args, PyObject* kw)
{
    static char* kwlist[] = {(char*)"command", NULL};
    char *buffer;
    if (!PyArg_ParseTupleAndKeywords(args, kw, "s", kwlist, &buffer))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    return Py_BuildValue("s", controller->Command(buffer));
}

static PyObject* Original_Command(PyObject* self, PyObject* args, PyObject* kw)
{
    static char* kwlist[] = {(char*)"command", NULL};
    char *buffer;
    if (!PyArg_ParseTupleAndKeywords(args, kw, "s", kwlist, &buffer))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    return Py_BuildValue("s", controller->Original_Command(buffer));
}

static PyObject* Step_Print_Initial(PyObject* self, PyObject* args, PyObject* kw)
{
    static char* kwlist[] = {(char*)"head", (char*)"format", NULL};
    char *head;
    char *format;
    if (!PyArg_ParseTupleAndKeywords(args, kw, "ss", kwlist, &head, &format))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    controller->Step_Print_Initial(head, format);
    return Py_BuildValue("");
}

static PyObject* Step_Print(PyObject* self, PyObject* args, PyObject* kw)
{
    static char* kwlist[] = {(char*)"head", (char*)"content", NULL};
    char *head;
    char *content;
    if (!PyArg_ParseTupleAndKeywords(args, kw, "ss", kwlist, &head, &content))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    controller->Step_Print(head, content);
    return Py_BuildValue("");
}

static PyObject* CV_Command_Exist(PyObject* self, PyObject* args, PyObject* kw)
{
    static char* kwlist[] = {(char*)"command", NULL};
    char *buffer;
    if (!PyArg_ParseTupleAndKeywords(args, kw, "s", kwlist, &buffer))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    if (cv_controller->Command_Exist(buffer))
        Py_RETURN_TRUE;
    else
        Py_RETURN_FALSE;
}

static PyObject* CV_Command(PyObject* self, PyObject* args, PyObject* kw)
{
    static char* kwlist[] = {(char*)"command", NULL};
    char *buffer;
    if (!PyArg_ParseTupleAndKeywords(args, kw, "s", kwlist, &buffer))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    return Py_BuildValue("s", cv_controller->Command(buffer));
}

static PyObject* CV_Original_Command(PyObject* self, PyObject* args, PyObject* kw)
{
    static char* kwlist[] = {(char*)"command", NULL};
    char *buffer;
    if (!PyArg_ParseTupleAndKeywords(args, kw, "s", kwlist, &buffer))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    return Py_BuildValue("s", cv_controller->Original_Command(buffer));
}

static PyObject* Get_CV(PyObject* self, PyObject* args, PyObject* kw)
{
    static char* kwlist[] = {(char*)"cv_name", NULL};
    char *buffer;
    if (!PyArg_ParseTupleAndKeywords(args, kw, "s", kwlist, &buffer))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    return Py_BuildValue("L", (intptr_t)cv_controller->get_CV(buffer));
}

struct CV_PYTHON: public COLLECTIVE_VARIABLE_PROTOTYPE
{
    void Initial(COLLECTIVE_VARIABLE_CONTROLLER* manager, int atom_numbers, const char* module_name);
    void Compute(int atom_numbers, UNSIGNED_INT_VECTOR* uint_crd, VECTOR scaler, VECTOR* crd, VECTOR box_length, int need, int step);
};

void CV_PYTHON::Initial(COLLECTIVE_VARIABLE_CONTROLLER* manager, int atom_numbers, const char* module_name)
{
    Super_Initial(manager, atom_numbers, module_name);
    char buffer[CHAR_LENGTH_MAX];
    sprintf(buffer, "Sponge.COLLECTIVE_VARIABLE_PROTOTYPE.New(%ld, '%s')", (intptr_t)this ,module_name); 
    PyRun_SimpleString(buffer);
}

void CV_PYTHON::Compute(int atom_numbers, UNSIGNED_INT_VECTOR* uint_crd, VECTOR scaler, VECTOR* crd, VECTOR box_length, int need, int step)
{
    char buffer[CHAR_LENGTH_MAX + 82];
    sprintf(buffer, "Sponge.COLLECTIVE_VARIABLE_PROTOTYPE.objects['%s'].Compute(%d, %d)", module_name, need, step); 
    PyRun_SimpleString(buffer);
}

static PyObject* Register_CV(PyObject* self, PyObject* args, PyObject* kw)
{
    static char* kwlist[] = {(char*)"type_name", NULL};
    char *buffer;
    if (!PyArg_ParseTupleAndKeywords(args, kw, "s", kwlist, &buffer))
    {
        PyErr_SetString(PyExc_Exception, "arguments are not right.");
        return NULL;
    }
    auto f = [buffer](COLLECTIVE_VARIABLE_CONTROLLER* manager, const char* name)
    {
        COLLECTIVE_VARIABLE_PROTOTYPE *cv = new CV_PYTHON;
        strcpy(cv->type_name, buffer);
        cv->Initial(manager, manager->atom_numbers, name);
        return cv;
    };
    CV_MAP[0][buffer] = f;
    return Py_BuildValue("");
}


static PyMethodDef SpongeMethod[] =
{
      {"_nl_atom_numbers",(PyCFunction)Neighbor_List_Atom_Numbers, METH_VARARGS, ""},
      {"_nl_atom_serials",(PyCFunction)Neighbor_List_Atom_Serials, METH_VARARGS, ""},
      {"_atom_numbers", (PyCFunction)Atom_Numbers, METH_VARARGS, ""},
      {"_crd", (PyCFunction)Coordinate, METH_VARARGS, ""},
      {"_vel", (PyCFunction)Velocity, METH_VARARGS, ""},
      {"_acc", (PyCFunction)Acceleration, METH_VARARGS, ""},
      {"_frc", (PyCFunction)Force, METH_VARARGS, ""},
      {"_d_mass", (PyCFunction)Mass, METH_VARARGS, ""},
      {"_d_charge", (PyCFunction)Charge, METH_VARARGS, ""},
      {"_d_atom_energy", (PyCFunction)Atom_Energy, METH_VARARGS, ""},
      {"_d_atom_virial", (PyCFunction)Atom_Virial, METH_VARARGS, ""},
      {"_steps", (PyCFunction)Steps, METH_VARARGS, ""},
      {"_box_length", (PyCFunction)Box_Length, METH_VARARGS, ""},
      {"_working_device", (PyCFunction)Working_Device, METH_VARARGS|METH_KEYWORDS, "" },
      {"_printf", (PyCFunction)Control_Printf, METH_VARARGS|METH_KEYWORDS, "" },
      {"_warn", (PyCFunction)Control_Warn, METH_VARARGS|METH_KEYWORDS, "" },
      {"_error", (PyCFunction)Control_Error, METH_VARARGS|METH_KEYWORDS, "" },
      {"_command_exist", (PyCFunction)Command_Exist, METH_VARARGS|METH_KEYWORDS, "" },
      {"_command", (PyCFunction)Command, METH_VARARGS|METH_KEYWORDS, "" },
      {"_original_command", (PyCFunction)Original_Command, METH_VARARGS|METH_KEYWORDS, "" },
      {"_step_print_initial", (PyCFunction)Step_Print_Initial, METH_VARARGS|METH_KEYWORDS, "" },
      {"_step_print", (PyCFunction)Step_Print, METH_VARARGS|METH_KEYWORDS, "" },
      {"_cv_command_exist", (PyCFunction)CV_Command_Exist, METH_VARARGS|METH_KEYWORDS, "" },
      {"_cv_command", (PyCFunction)CV_Command, METH_VARARGS|METH_KEYWORDS, "" },
      {"_cv_original_command", (PyCFunction)CV_Original_Command, METH_VARARGS|METH_KEYWORDS, "" },
      {"_get_cv", (PyCFunction)Get_CV, METH_VARARGS|METH_KEYWORDS, "" },
      {"_register_cv", (PyCFunction)Register_CV, METH_VARARGS|METH_KEYWORDS, "" },
      {NULL,NULL,0,NULL}
};

static PyModuleDef SpongeModule = 
{
  PyModuleDef_HEAD_INIT, "Sponge", NULL, -1, SpongeMethod,
  NULL, NULL, NULL, NULL
};

PyMODINIT_FUNC PyInit_sponge(void)
{
    PyObject *m;
    if (PyType_Ready(&SpongeCVType) < 0)
        return NULL;
    m = PyModule_Create(&SpongeModule);
    if (m == NULL)
        return NULL;
    Py_INCREF(&SpongeCVType);
    if (PyModule_AddObject(m, "_COLLECTIVE_VARIABLE_PROTOTYPE", (PyObject *) &SpongeCVType) < 0) 
    {
        Py_DECREF(&SpongeCVType);
        Py_DECREF(m);
        return NULL;
    }
    return m;
}

PLUGIN_API std::string Name()
{
    return std::string("Python Runtime Interface Plugin");
}

PLUGIN_API std::string Version()
{
    return std::string("1.4b0");
}

PLUGIN_API std::string Version_Check(int i)
{
    if (i < 20231219)
    {
        return std::string("Reason:\n\tPRIPS v1.4b0 can not be used with SPONGE before 20231219. Your SPONGE: " + std::to_string(i));
    }
    return std::string();
}

PLUGIN_API void Initial(MD_INFORMATION* md, CONTROLLER* ctrl, NEIGHBOR_LIST* nl,
                        COLLECTIVE_VARIABLE_CONTROLLER* cv, CV_MAP_TYPE* cv_map, CV_INSTANCE_TYPE* cv_instance_map)
{
    md_info = md;
    controller = ctrl;
    neighbor_list = nl;
    cv_controller = cv;
    CV_MAP = cv_map;
    CV_INSTANCE_MAP = cv_instance_map;
    controller->printf("    initializing pyplugin\n");
    if (!controller->Command_Exist("py"))
    {
        controller->printf("        No 'py' command found. Pyplugin will not be initialized.\n");
        return;
    }
    PyImport_AppendInittab("Sponge",&PyInit_sponge);
    Py_Initialize();
    if (!Py_IsInitialized())
    {
        fprintf(stderr, "        Python Initialize Failed.\n");
        getchar();
        exit(1);
    }
    else
    {
        controller->printf("        Python Initialized\n");
    }
    wchar_t *temp_args[1] = {(wchar_t *)L"SPONGE"};
    PySys_SetArgv(1, temp_args); 
    PyRun_SimpleString(R"XYJ(
import sys
sys.dont_write_bytecode = True
from pathlib import Path
import importlib.util as iu
import cupy as cp
from enum import Enum, Flag
import Sponge

old_excepthook = sys.excepthook
def new_hook(exctype, value, traceback):
    old_excepthook(exctype, value, traceback)
    exit(1)
sys.excepthook = new_hook
del new_hook

def _get_cupy_array(tuple):
    return cp.ndarray(tuple[1], dtype=tuple[2], memptr=cp.cuda.MemoryPointer(cp.cuda.UnownedMemory(*tuple[0]), 0))
Sponge._get_cupy_array = _get_cupy_array

class spongeError(Enum):
    spongeSuccess = 0
    # 未实现的功能
    spongeErrorNotImplemented = 1001
    # 文件格式（编码、换行符）问题 或 数据格式不正确
    spongeErrorBadFileFormat = 1002
    # 冲突的命令
    spongeErrorConflictingCommand = 1003
    # 缺失的命令
    spongeErrorMissingCommand = 1004
    # 类型错误的命令
    spongeErrorTypeErrorCommand = 1005
    # 值错误的命令
    spongeErrorValueErrorCommand = 1006
    # 模拟崩溃
    spongeErrorSimulationBreakDown = 1007
    # 内存分配失败
    spongeErrorMallocFailed = 1008
    # 越界
    spongeErrorOverflow = 1009
    # 打开文件失败
    spongeErrorOpenFileFailed = 1010

Sponge.spongeError = spongeError
del spongeError

class CONTROLLER:
    """
        this **class** is the module to control the input, output and cuda in SPONGE
    """
    def __init__(self):
        self.formats = {}

    @property
    def working_device(self):
        """ The working device id """
        return Sponge._working_device()

    def printf(self, *values, sep=" ", end="\n"):
        """ Print the values to the screen and the mdinfo file """
        return Sponge._printf(sep.join([f"{i}" for i in values]) + end)

    def Warn(self, warning):
        """ Print the warning to the screen and the mdinfo file """ 
        return Sponge._warn(warning)

    def Throw_SPONGE_Error(self, error, error_by, error_hint):
        """ Raise error to the screen and the mdinfo file, then exit """
        error = Sponge.spongeError(error)
        Sponge._error(error.value, error_by, error_hint)

    def Command_Exist(self, arg1, arg2=None):
        """
        Check the command whether exist in SPONGE.
        If Only arg1 is given, the command to check is "arg1".
        If arg1 and arg2 is given, the command to check is "arg1_arg2".

        :param arg1: the command or the command prefix
        :param arg2: the command suffix
        :return: True if the command exist, else False
        """
        if arg2 is None:
            return Sponge._command_exist(arg1)
        else:
            return Sponge._command_exist(f"{arg1}_{arg2}")

    def Command(self, arg1, arg2=None):
        """
        Get the simplified value of the command in SPONGE.
        If Only arg1 is given, the command to get the value is "arg1".
        If arg1 and arg2 is given, the command to get the value is "arg1_arg2".

        :param arg1: the command or the command prefix
        :param arg2: the command suffix
        :return: a string, the simplified value of the command
        """
        if arg2 is None:
            return Sponge._command(arg1)
        else:
            return Sponge._command(f"{arg1}_{arg2}")

    def Original_Command(self, arg1, arg2=None):
        """
        Get the original value of the command in SPONGE.
        If Only arg1 is given, the command to get the value is "arg1".
        If arg1 and arg2 is given, the command to get the value is "arg1_arg2".

        :param arg1: the command or the command prefix
        :param arg2: the command suffix
        :return: a string, the original value of the command
        """
        if arg2 is None:
            return Sponge._original_command(arg1)
        else:
            return Sponge._original_command(f"{arg1}_{arg2}")

    def Step_Print_Initial(self, head, format):
        """
        Initialize a step print item. A step print item is an item to print in the screen and in the mdout file.

        :param head: the name of the item
        :param format: the format of the step print item
        """
        self.formats[head] = format
        Sponge._step_print_initial(head, "%s")

    def Step_Print(self, head, content):
        """
        Print a step print item. A step print item is an item to print in the screen and in the mdout file.

        :param head: the name of the item
        :param content: the content of the step print item
        """
        Sponge._step_print(head, self.formats[head]%content)

    def __repr__(self):
        return "< IO and CUDA controller of SPONGE >"

Sponge.CONTROLLER = CONTROLLER
Sponge.controller = CONTROLLER()
del CONTROLLER

class COLLECTIVE_VARIABLE_NEED(Flag):
    CV_NEED_NONE = 0
    CV_NEED_GPU_VALUE = 1
    CV_NEED_CRD_GRADS = 2
    CV_NEED_CPU_VALUE = 4
    CV_NEED_BOX_GRADS = 8
    CV_NEED_ALL = 15

Sponge.COLLECTIVE_VARIABLE_NEED = COLLECTIVE_VARIABLE_NEED
del COLLECTIVE_VARIABLE_NEED

class COLLECTIVE_VARIABLE_PROTOTYPE:
    types = {}
    objects = {}
    def __init_subclass__(cls, type_name=None):
        if type_name is None:
            type_name = cls.__name__
        Sponge.COLLECTIVE_VARIABLE_PROTOTYPE.types[type_name] = cls
        Sponge._register_cv(type_name)

    def __init__(self, memptr):
        self._cv = Sponge._COLLECTIVE_VARIABLE_PROTOTYPE(memptr)
        self._cuda_stream = cp.cuda.ExternalStream(self._cv._get_cuda_stream(), Sponge.controller.working_device)
        self._d_value = Sponge._get_cupy_array(self._cv._d_value())
        self._crd_grads = Sponge._get_cupy_array(self._cv._crd_grads())
        self._box_grads = Sponge._get_cupy_array(self._cv._box_grads())

    @property
    def d_value(self):
        return self._d_value
    
    @d_value.setter
    def d_value(self, value):
        self._d_value[:] = value

    @property
    def crd_grads(self):
        return self._crd_grads

    @crd_grads.setter
    def crd_grads(self, value):
        self._crd_grads[:] = value

    @property
    def box_grads(self):
        return self._box_grads

    @box_grads.setter
    def box_grads(self, value):
        self._box_grads[:] = value

    @property
    def hip_stream(self):
        return self._cuda_stream

    def get_last_update_step(self, need):
        return self._cv._get_last_update_step(need)
    
    def set_last_update_step(self, need, step):
        self._cv._set_last_update_step(need, step)

    def Check_Whether_Computed_At_This_Step(self, step, need):
        if (need & Sponge.CV_NEED_CPU_VALUE) and self.get_last_update_step(Sponge.CV_NEED_CPU_VALUE) == step:
            need &= ~Sponge.CV_NEED_CPU_VALUE
        if (need & Sponge.CV_NEED_GPU_VALUE) and self.get_last_update_step(Sponge.CV_NEED_GPU_VALUE) == step:
            need &= ~Sponge.CV_NEED_GPU_VALUE
        if (need & Sponge.CV_NEED_CRD_GRADS) and self.get_last_update_step(Sponge.CV_NEED_CRD_GRADS) == step:
            need &= ~Sponge.CV_NEED_CRD_GRADS
        if (need & Sponge.CV_NEED_BOX_GRADS) and self.get_last_update_step(Sponge.CV_NEED_BOX_GRADS) == step:
            need &= ~Sponge.CV_NEED_BOX_GRADS
        return need

    @classmethod
    def New(cls, memptr, module_name):
        type_name = Sponge.cv_controller.Command(module_name, "type")
        cls.objects[module_name] = cls.types[type_name](memptr, module_name)

    @property
    def value(self):
        return self._cv._get_value()
    
    @value.setter
    def value(self, value):
        return self._cv._set_value(value)


Sponge.COLLECTIVE_VARIABLE_PROTOTYPE = COLLECTIVE_VARIABLE_PROTOTYPE
del COLLECTIVE_VARIABLE_PROTOTYPE

class COLLECTIVE_VARIABLE_CONTROLLER(Sponge.CONTROLLER):
    def Get_CV(self, cv_name):
        return Sponge.COLLECTIVE_VARIABLE_PROTOTYPE(Sponge._get_cv(cv_name))

    def Command_Exist(self, arg1, arg2=None):
        if arg2 is None:
            return Sponge._cv_command_exist(arg1)
        else:
            return Sponge._cv_command_exist(f"{arg1}_{arg2}")

    def Command(self, arg1, arg2=None):
        if arg2 is None:
            return Sponge._cv_command(arg1)
        else:
            return Sponge._cv_command(f"{arg1}_{arg2}")

    def Original_Command(self, arg1, arg2=None):
        if arg2 is None:
            return Sponge._cv_original_command(arg1)
        else:
            return Sponge._cv_original_command(f"{arg1}_{arg2}")

    def Ask_For_CV(self, name, N, verbose_level=0):
        if self.Command_Exist(name, "CV"):
            cvs = self.Original_Command(name, "CV").split()
        if N > 0 and len(cvs) != N:
            self.printf("    Error: %d CV(s) should be given to %s, but %d found\n"%(N, name, len(cvs)))
            input()
            exit(1)
        elif N <= 0 and len(cvs) < -N:
            self.printf("    Error: at least %d CV(s) should be given to %s, but only %d found"%(N, name, len(cvs)))
            input()
            exit(1)
        if verbose_level > -2:
            self.printf("    %d CV(s) found for %s\n"%(len(cvs), name));
        to_ret = []
        for cv_name in cvs:
            if verbose_level > -1:
                self.printf("        CV %d: %s\n"%(i, cv_name))
            to_ret.append(get_CV(cv_name))
            if verbose_level > -1:
                self.printf("        type of '%s' is '%s'\n"%(cv_name, to_ret[-1].type_name));

    def __repr__(self):
        return "< cv controller of SPONGE>"

Sponge.COLLECTIVE_VARIABLE_CONTROLLER = COLLECTIVE_VARIABLE_CONTROLLER
Sponge.cv_controller = COLLECTIVE_VARIABLE_CONTROLLER()
del COLLECTIVE_VARIABLE_CONTROLLER

class MD_INFORMATION:
    """ Contains the information for MD simulations """
    def __init__(self):
        self._crd = _get_cupy_array(Sponge._crd())
        self._vel = _get_cupy_array(Sponge._vel())
        self._frc = _get_cupy_array(Sponge._frc())
        self._d_mass = _get_cupy_array(Sponge._d_mass())
        self._d_charge = _get_cupy_array(Sponge._d_charge())
        self._d_atom_energy = _get_cupy_array(Sponge._d_atom_energy())
        self._d_atom_virial = _get_cupy_array(Sponge._d_atom_virial())

        class system_information:
            """ the system information of MD information """
            @property
            def steps(self):
                """ the current step of the simulation """
                return Sponge._steps()
            @property
            def box_length(self):
                """ the length of the box """
                return Sponge._box_length()

        self._sys = system_information()

    @property
    def atom_numbers(self):
        """ the total number of atoms """
        return Sponge._atom_numbers()

    @property
    def crd(self):
        """ the position coordinates of the atoms """
        return self._crd

    @crd.setter
    def crd(self, value):
        self._crd[:] = value

    @property
    def vel(self):
        """ the velocity of the atoms """
        return self._vel

    @vel.setter
    def vel(self, value):
        self._vel[:] = value

    @property
    def frc(self):
        """ the force of the atoms """
        return self._frc

    @frc.setter
    def frc(self, value):
        self._frc[:] = value

    @property
    def d_mass(self):
        """ the mass of the atoms """
        return self._d_mass

    @property
    def d_charge(self):
        """ the charge of the atoms (In the unit of SPONGE, a unit charge is 18.2223) """
        return self._d_charge

    @d_charge.setter
    def d_charge(self, value):
        self._d_charge[:] = value

    @property
    def d_atom_energy(self):
        """ the energy of the atoms """
        return self._d_atom_energy

    @d_atom_energy.setter
    def d_atom_energy(self, value):
        self._d_atom_energy[:] = value

    @property
    def d_atom_virial(self):
        """ the virial of the atoms """
        return self._d_atom_virial

    @d_atom_virial.setter
    def d_atom_virial(self, value):
        self._d_atom_virial[:] = value

    @property
    def sys(self):
        """ the system information """
        return self._sys

    def __repr__(self):
        return "< MD information container of SPONGE >"

Sponge.MD_INFORMATION = MD_INFORMATION
Sponge.md_info = MD_INFORMATION()
del MD_INFORMATION

class NEIGHBOR_LIST:
    """ 
        Neighbor List of SPONGE.
        This class should be initialized at After_Initial, because neighbor_list is not initialized when initializing prips"""
    def __init__(self):
        self._atom_numbers = Sponge._get_cupy_array(Sponge._nl_atom_numbers())
        self._atom_serials = Sponge._get_cupy_array(Sponge._nl_atom_serials())

    @property
    def atom_numbers(self):
        """ the number of neighbors for every atom """
        return self._atom_numbers[:, 0]

    @property
    def atom_serials(self):
        """ the serial of neighbors for every atom """
        return self._atom_serials

Sponge.NEIGHBOR_LIST = NEIGHBOR_LIST
del NEIGHBOR_LIST

del _get_cupy_array
    )XYJ");
    char buffer[CHAR_LENGTH_MAX];
    sprintf(buffer, "Sponge.fname = r'%s'", controller->Command("py")); 
    PyRun_SimpleString(buffer);
    PyRun_SimpleString(R"XYJ(sponge_pyplugin_path = Path(Sponge.fname)
spec = iu.spec_from_file_location('sponge_pyplugin', sponge_pyplugin_path)
sponge_pyplugin = iu.module_from_spec(spec)
spec.loader.exec_module(sponge_pyplugin)
Sponge.controller.printf("        module '%s' imported."%(sponge_pyplugin_path.stem))
    )XYJ");
    is_initialized = 1;
    controller->printf("    end initializing pyplugin\n");
}

PLUGIN_API void After_Initial()
{
    if (!is_initialized)
        return;
    PyRun_SimpleString(R"XYJ(
if hasattr(sponge_pyplugin, "After_Initial"):
    sponge_pyplugin.After_Initial()
    )XYJ");
}

PLUGIN_API void Calculate_Force()
{
    if (!is_initialized)
        return;
    PyRun_SimpleString(R"XYJ(
if hasattr(sponge_pyplugin, "Calculate_Force"):
    sponge_pyplugin.Calculate_Force()
    )XYJ");
}

PLUGIN_API void Mdout_Print()
{
    if (!is_initialized)
        return;
    PyRun_SimpleString(R"XYJ(
if hasattr(sponge_pyplugin, "Mdout_Print"):
    sponge_pyplugin.Mdout_Print()
    )XYJ");
}
