#include "hip/hip_runtime.h"
﻿#include "MD_core.cuh"
#include "thrust/sort.h"

#define BOX_TRAJ_COMMAND "box"
#define BOX_TRAJ_DEFAULT_FILENAME "mdbox.txt"
#define TRAJ_COMMAND "crd"
#define TRAJ_DEFAULT_FILENAME "mdcrd.dat"
#define RESTART_COMMAND "rst"
#define RESTART_DEFAULT_FILENAME "restart"
//20210827用于输出速度和力
#define FRC_TRAJ_COMMAND "frc"
#define VEL_TRAJ_COMMAND "vel"

//20230303 用于错误输出原因
#define ATOM_NUMBERS_DISMATCH "Reason:\n\t'atom_numbers' (the number of atoms) is diiferent in different input files\n"
#define ATOM_NUMBERS_MISSING "Reason:\n\tno 'atom_numbers' (the number of atoms) found. No 'mass_in_file' or 'amber_parm7' is provided\n"


//用来将原子的真实坐标转换为unsigned int坐标,注意factor需要乘以0.5（保证越界坐标自然映回box）
static __global__ void Crd_To_Uint_Crd(const int atom_numbers, const VECTOR box_length_inverse, const VECTOR *crd, UNSIGNED_INT_VECTOR *uint_crd)
{
    int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
    if (atom_i < atom_numbers)
    {
        UNSIGNED_INT_VECTOR uinti;
        VECTOR temp = crd[atom_i];

        temp.x *= box_length_inverse.x;
        temp.y *= box_length_inverse.y;
        temp.z *= box_length_inverse.z;

        temp.x -= floorf(temp.x);
        temp.y -= floorf(temp.y);
        temp.z -= floorf(temp.z);

        uinti.uint_x = CONSTANT_UINT_MAX_FLOAT * temp.x;
        uinti.uint_y = CONSTANT_UINT_MAX_FLOAT * temp.y;
        uinti.uint_z = CONSTANT_UINT_MAX_FLOAT * temp.z;

        uint_crd[atom_i] = uinti;
    }
}

static __global__ void MD_Iteration_Leap_Frog
(const int atom_numbers, VECTOR *vel, VECTOR *crd, VECTOR *frc, VECTOR *acc, const float *inverse_mass, const float dt)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        acc[i].x = inverse_mass[i] * frc[i].x;
        acc[i].y = inverse_mass[i] * frc[i].y;
        acc[i].z = inverse_mass[i] * frc[i].z;

        vel[i].x = vel[i].x + dt*acc[i].x;
        vel[i].y = vel[i].y + dt*acc[i].y;
        vel[i].z = vel[i].z + dt*acc[i].z;

        crd[i].x = crd[i].x + dt*vel[i].x;
        crd[i].y = crd[i].y + dt*vel[i].y;
        crd[i].z = crd[i].z + dt*vel[i].z;

    }
}

static __global__ void Scale_Position_To_Center_CUDA(const int atom_numbers, const float scaler, VECTOR* crd, VECTOR half_box_length)
{
    int atom_i = blockIdx.x * blockDim.x + threadIdx.x;
    if (atom_i < atom_numbers)
    {
        crd[atom_i] = scaler * (crd[atom_i] - half_box_length) + half_box_length;   
    }
}

static __global__ void MD_Iteration_Leap_Frog_With_Max_Velocity
(const int atom_numbers, VECTOR *vel, VECTOR *crd, VECTOR *frc, VECTOR *acc, const float *inverse_mass, const float dt, const float max_velocity)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        VECTOR acc_i = inverse_mass[i] * frc[i];
        VECTOR vel_i = vel[i] + dt * acc_i;
        vel_i = Make_Vector_Not_Exceed_Value(vel_i, max_velocity);
        vel[i] = vel_i;
        crd[i] = crd[i] + dt * vel_i;
    }
}

static __global__ void MD_Iteration_Gradient_Descent
(const int atom_numbers, VECTOR *crd, VECTOR *frc, const float *mass_inverse, const float dt, VECTOR *vel, const float momentum_keep)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        vel[i] = momentum_keep * vel[i] + dt * mass_inverse[i] * frc[i];
        crd[i] = crd[i] + dt * vel[i];
    }
}

static __global__ void MD_Iteration_Gradient_Descent_With_Max_Move
(const int atom_numbers, VECTOR *crd, VECTOR *frc, const float *mass_inverse, const float dt, VECTOR *vel, const float momentum_keep, float max_move)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        vel[i] = momentum_keep * vel[i] + dt * mass_inverse[i] * frc[i];
        VECTOR move = dt * vel[i];
        move = Make_Vector_Not_Exceed_Value(move, max_move);
        crd[i] = crd[i] + move;
    }
}

static __global__ void MD_Iteration_Speed_Verlet_1(const int atom_numbers, const float half_dt, const float dt, const VECTOR *acc, VECTOR *vel, VECTOR *crd, VECTOR *frc)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        vel[i].x = vel[i].x + half_dt*acc[i].x;
        vel[i].y = vel[i].y + half_dt*acc[i].y;
        vel[i].z = vel[i].z + half_dt*acc[i].z;
        crd[i].x = crd[i].x + dt*vel[i].x;
        crd[i].y = crd[i].y + dt*vel[i].y;
        crd[i].z = crd[i].z + dt*vel[i].z;
    }
}

static __global__ void MD_Iteration_Speed_Verlet_2(const int atom_numbers, const float half_dt, const float *inverse_mass, const VECTOR *frc, VECTOR *vel, VECTOR *acc)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        acc[i].x = inverse_mass[i] * frc[i].x;
        acc[i].y = inverse_mass[i] * frc[i].y;
        acc[i].z = inverse_mass[i] * frc[i].z;
        vel[i].x = vel[i].x + half_dt*acc[i].x;
        vel[i].y = vel[i].y + half_dt*acc[i].y;
        vel[i].z = vel[i].z + half_dt*acc[i].z;
    }
}

static __global__ void MD_Iteration_Speed_Verlet_2_With_Max_Velocity(const int atom_numbers, const float half_dt, const float *inverse_mass, const VECTOR *frc, VECTOR *vel, VECTOR *acc, const float max_velocity)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        VECTOR acc_i = inverse_mass[i] * frc[i];
        VECTOR vel_i = vel[i] + half_dt * acc_i;

        vel[i] = Make_Vector_Not_Exceed_Value(vel_i, max_velocity);
        acc[i] = acc_i;
    }
}

static __global__ void Get_Origin(const int residue_numbers, const int *start, const int *end,
    const VECTOR *crd, const float *atom_mass, const float *residue_mass_inverse, VECTOR *center_of_mass)
{
    int residue_i = blockDim.x * blockIdx.x + threadIdx.x;
    if (residue_i < residue_numbers)
    {
        VECTOR origin = { FLT_MAX, FLT_MAX, FLT_MAX };
        for (int atom_i = start[residue_i]; atom_i < end[residue_i]; atom_i += 1)
        {
            VECTOR crd_i = crd[atom_i]; 
            origin.x = fminf(origin.x, crd_i.x);
            origin.y = fminf(origin.y, crd_i.y);
            origin.z = fminf(origin.z, crd_i.z);
        }
        center_of_mass[residue_i] = origin;
    }
}

static __global__ void Map_Center_Of_Mass(const int residue_numbers, const int *start, const int *end,
    const float scaler, const VECTOR *center_of_mass, const VECTOR box_length, VECTOR *crd, int* periodicity)
{
    VECTOR trans_vec;
    VECTOR com;
    int residue_i = blockDim.x * blockIdx.x + threadIdx.x;
    if (residue_i < residue_numbers)
    {
        if (periodicity == NULL || periodicity[residue_i] == 0)
        {
            com = center_of_mass[residue_i];
            trans_vec.x = com.x - floorf(com.x / box_length.x) * box_length.x;
            trans_vec.y = com.y - floorf(com.y / box_length.y) * box_length.y;
            trans_vec.z = com.z - floorf(com.z / box_length.z) * box_length.z;
            trans_vec.x = scaler * trans_vec.x - com.x;
            trans_vec.y = scaler * trans_vec.y - com.y;
            trans_vec.z = scaler * trans_vec.z - com.z;
            for (int atom_i = start[residue_i] + threadIdx.y; atom_i < end[residue_i]; atom_i += blockDim.y)
            {
                crd[atom_i] = crd[atom_i] + trans_vec;
            }
        }
        else
        {
            for (int atom_i = start[residue_i] + threadIdx.y; atom_i < end[residue_i]; atom_i += blockDim.y)
            {
                com = crd[atom_i];
                trans_vec.x = com.x - floorf(com.x / box_length.x) * box_length.x;
                trans_vec.y = com.y - floorf(com.y / box_length.y) * box_length.y;
                trans_vec.z = com.z - floorf(com.z / box_length.z) * box_length.z;
                trans_vec.x = scaler * trans_vec.x - com.x;
                trans_vec.y = scaler * trans_vec.y - com.y;
                trans_vec.z = scaler * trans_vec.z - com.z;
                crd[atom_i] = crd[atom_i] + trans_vec;
            }
        }
    }
}

static __global__ void Map_Center_Of_Mass(const int residue_numbers, const int *start, const int *end,
    const VECTOR scaler, const VECTOR *center_of_mass, const VECTOR box_length, VECTOR *crd, int* periodicity)
{
    VECTOR trans_vec;
    VECTOR com;
    int residue_i = blockDim.x * blockIdx.x + threadIdx.x;
    if (residue_i < residue_numbers)
    {
        if (periodicity == NULL || periodicity[residue_i] == 0)
        {
            com = center_of_mass[residue_i];
            trans_vec.x = com.x - floorf(com.x / box_length.x) * box_length.x;
            trans_vec.y = com.y - floorf(com.y / box_length.y) * box_length.y;
            trans_vec.z = com.z - floorf(com.z / box_length.z) * box_length.z;
            trans_vec.x = scaler.x * trans_vec.x - com.x;
            trans_vec.y = scaler.y * trans_vec.y - com.y;
            trans_vec.z = scaler.z * trans_vec.z - com.z;
            for (int atom_i = start[residue_i] + threadIdx.y; atom_i < end[residue_i]; atom_i += blockDim.y)
            {
                crd[atom_i] = crd[atom_i] + trans_vec;
            }
        }
        else
        {
            for (int atom_i = start[residue_i] + threadIdx.y; atom_i < end[residue_i]; atom_i += blockDim.y)
            {
                com = crd[atom_i];
                trans_vec.x = com.x - floorf(com.x / box_length.x) * box_length.x;
                trans_vec.y = com.y - floorf(com.y / box_length.y) * box_length.y;
                trans_vec.z = com.z - floorf(com.z / box_length.z) * box_length.z;
                trans_vec.x = scaler.x * trans_vec.x - com.x;
                trans_vec.y = scaler.y * trans_vec.y - com.y;
                trans_vec.z = scaler.z * trans_vec.z - com.z;
                crd[atom_i] = crd[atom_i] + trans_vec;
            }
        }
    }
}

static __global__ void Add_Sum_List(int n, float *atom_virial, float *sum_virial)
{
    float temp = 0;
    for (int i = threadIdx.x; i < n; i = i + blockDim.x)
    {
        temp = temp + atom_virial[i];
    }
    atomicAdd(sum_virial, temp);
}

static __global__ void Calculate_Pressure_Cuda(const float V_inverse, const float *ek, const float *virial, float *pressure)
{
    pressure[0] = (ek[0] * 2 + virial[0]) * 0.33333333333333f * V_inverse;
}

static __global__ void MD_Residue_Ek
(const int residue_numbers,const int *start,const int *end,float *ek,
const VECTOR *atom_vel,const float *atom_mass)
{
    int residue_i = blockDim.x*blockIdx.x + threadIdx.x;
    if (residue_i < residue_numbers)
    {
        VECTOR momentum = {0.,0.,0.};
        float res_mass = 0.; //待提出，只需要初始时计算一遍
        int s = start[residue_i];
        int e = end[residue_i];
        float mass_lin;
        for (int atom_i = s; atom_i < e; atom_i = atom_i + 1)
        {
            mass_lin = atom_mass[atom_i];

            momentum.x = momentum.x + mass_lin*atom_vel[atom_i].x;
            momentum.y = momentum.y + mass_lin*atom_vel[atom_i].y;
            momentum.z = momentum.z + mass_lin*atom_vel[atom_i].z;
            res_mass = res_mass + mass_lin;
        }
        ek[residue_i] = 0.5*(momentum.x*momentum.x + momentum.y*momentum.y + momentum.z*momentum.z) / res_mass;
    }
}

static __global__ void MD_Atom_Ek
(const int atom_numbers, float *ek, const VECTOR *atom_vel, const float *atom_mass)
{
    int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
    if (atom_i < atom_numbers)
    {
        VECTOR v = atom_vel[atom_i];
        ek[atom_i] = 0.5 * v * v * atom_mass[atom_i];
    }
}

void MD_INFORMATION::system_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    this->md_info = md_info;
    steps = 0;
    if (md_info->mode != md_info->RERUN)
    {
        step_limit = 1000;
        if (controller[0].Command_Exist("step_limit"))
        {
            controller->Check_Int("step_limit", "MD_INFORMATION::system_information::Initial");
            step_limit = atoi(controller[0].Command("step_limit"));
        }

        target_temperature = 300.0f;
        if (md_info->mode >= md_info->NVT && controller[0].Command_Exist("target_temperature"))
        {
            controller->Check_Float("target_temperature", "MD_INFORMATION::system_information::Initial");
            target_temperature = atof(controller[0].Command("target_temperature"));
        }

        target_pressure = 1;
        if (md_info->mode == md_info->NPT && controller[0].Command_Exist("target_pressure"))
        {
            controller->Check_Float("target_pressure", "MD_INFORMATION::system_information::Initial");
            target_pressure = atof(controller[0].Command("target_pressure"));
        }
        target_pressure *= CONSTANT_PRES_CONVERTION_INVERSE;
    }
    else
    {
        step_limit = INT_MAX - 1;
        if (controller[0].Command_Exist("frame_limit"))
        {
            controller->Check_Int("frame_limit", "MD_INFORMATION::system_information::Initial");
            step_limit = atoi(controller[0].Command("frame_limit"));
        }
    }
    for (int i = 0; i < md_info->atom_numbers; i++)
    {
        std::set<int> temp;
        connectivity[i] = temp;
    }
    if (md_info->mode != md_info->RERUN)
    {
        controller->Step_Print_Initial("step", "%d");
        controller->Step_Print_Initial("time", "%.3lf");
        controller->Step_Print_Initial("temperature", "%.2f");
    }
    else
    {
        controller->Step_Print_Initial("frame", "%d");
    }
    Cuda_Malloc_Safely((void**)&this->d_virial, sizeof(float));
    Cuda_Malloc_Safely((void**)&this->d_pressure, sizeof(float));
    Cuda_Malloc_Safely((void**)&this->d_temperature, sizeof(float));
    Cuda_Malloc_Safely((void**)&this->d_potential, sizeof(float));
    Cuda_Malloc_Safely((void**)&this->d_sum_of_atom_ek, sizeof(float));
}

void MD_INFORMATION::non_bond_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    if (controller[0].Command_Exist("skin"))
    {
        controller->Check_Float("skin", "MD_INFORMATION::non_bond_information::Initial");
        skin = atof(controller[0].Command("skin"));
    }
    else
    {
        skin = 2.0;
    }
    controller->printf("    skin set to %.2f Angstrom\n", skin);

    if (controller[0].Command_Exist("cutoff"))
    {
        controller->Check_Float("cutoff", "MD_INFORMATION::non_bond_information::Initial");
        cutoff = atof(controller[0].Command("cutoff"));
    }
    else
    {
        cutoff = 10.0;
    }
    controller->printf("    cutoff set to %.2f Angstrom\n", cutoff);
    /*===========================
    读取排除表相关信息
    ============================*/
    if (controller[0].Command_Exist("exclude_in_file"))
    {
        FILE *fp = NULL;
        controller->printf("    Start reading excluded list:\n");
        Open_File_Safely(&fp, controller[0].Command("exclude_in_file"), "r");
        
        int atom_numbers = 0;
        int scanf_ret = fscanf(fp, "%d %d", &atom_numbers, &excluded_atom_numbers);
        if (scanf_ret != 2)
        {
            controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of exclude_in_file is not right\n");
        }
        if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
        {
            controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::non_bond_information::Initial", ATOM_NUMBERS_DISMATCH);
        }
        else if (md_info->atom_numbers == 0)
        {
            md_info->atom_numbers = atom_numbers;
        }
        controller->printf("        excluded list total length is %d\n", excluded_atom_numbers);

        Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
        Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);
        Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);

        Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
        Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
        Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);
        int count = 0;
        for (int i = 0; i < atom_numbers; i++)
        {
            scanf_ret = fscanf(fp, "%d", &h_excluded_numbers[i]);
            if (scanf_ret != 1)
            {
                controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "Reason:\n\tThe format of exclude_in_file is not right\n");
            }
            h_excluded_list_start[i] = count;
            for (int j = 0; j < h_excluded_numbers[i]; j++)
            {
                scanf_ret = fscanf(fp, "%d", &h_excluded_list[count]);
                if (scanf_ret != 1)
                {
                    controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "Reason:\n\tThe format of exclude_in_file is not right\n");
                }
                count++;
            }
        }
        if (count != excluded_atom_numbers)
        {
            controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", 
                "Reason:\n\tThe format of exclude_in_file is not right (excluded_atom_numbers is not right)\n");
        }
        hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
        controller->printf("    End reading excluded list\n\n");
        fclose(fp);
    }
    else if (controller[0].Command_Exist("amber_parm7"))
    {
        /*===========================
        从parm中读取排除表相关信息
        ============================*/
        FILE *parm = NULL;
        Open_File_Safely(&parm, controller[0].Command("amber_parm7"), "r");
        controller->printf("    Start reading excluded list from AMBER file:\n");
        while (true)
        {
            char temps[CHAR_LENGTH_MAX];
            char temp_first_str[CHAR_LENGTH_MAX];
            char temp_second_str[CHAR_LENGTH_MAX];
            if (!fgets(temps, CHAR_LENGTH_MAX, parm))
            {
                break;
            }
            if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
            {
                continue;
            }
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "POINTERS") == 0)
            {
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);

                int atom_numbers = 0;
                int scanf_ret = fscanf(parm, "%d\n", &atom_numbers);
                if (scanf_ret != 1)
                {
                    controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                }
                if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
                {
                    controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::non_bond_information::Initial", ATOM_NUMBERS_DISMATCH);
                }
                else if (md_info->atom_numbers == 0)
                {
                    md_info->atom_numbers = atom_numbers;
                }
                Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
                Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);

                Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
                Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
                for (int i = 0; i < 9; i = i + 1)
                {
                    scanf_ret = fscanf(parm, "%d\n", &excluded_atom_numbers);
                    if (scanf_ret != 1)
                    {
                        controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                    }
                }
                scanf_ret = fscanf(parm, "%d\n", &excluded_atom_numbers);
                if (scanf_ret != 1)
                {
                    controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                }
                controller->printf("        excluded list total length is %d\n", excluded_atom_numbers);

                Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);
                Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);
            }

            //read atom_excluded_number for every atom
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "NUMBER_EXCLUDED_ATOMS") == 0)
            {
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
                for (int i = 0; i<md_info->atom_numbers; i = i + 1)
                {
                    int scanf_ret = fscanf(parm, "%d\n", &h_excluded_numbers[i]);
                    if (scanf_ret != 1)
                    {
                        controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                    }
                }
            }
            //read every atom's excluded atom list
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "EXCLUDED_ATOMS_LIST") == 0)
            {
                int count = 0;
                int lin = 0;
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
                for (int i = 0; i<md_info->atom_numbers; i = i + 1)
                {
                    h_excluded_list_start[i] = count;
                    for (int j = 0; j<h_excluded_numbers[i]; j = j + 1)
                    {
                        int scanf_ret = fscanf(parm, "%d\n", &lin);
                        if (scanf_ret != 1)
                        {
                            controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                        }
                        if (lin == 0)
                        {
                            h_excluded_numbers[i] = 0;
                            break;
                        }
                        else
                        {
                            h_excluded_list[count] = lin - 1;
                            count = count + 1;
                        }
                    }
                    if (h_excluded_numbers[i] > 0)
                        thrust::sort(&h_excluded_list[h_excluded_list_start[i]], &h_excluded_list[h_excluded_list_start[i]] + h_excluded_numbers[i]);
                }
            }
        }

        hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*md_info->atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*md_info->atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
        controller->printf("    End reading excluded list from AMBER file\n\n");
        fclose(parm);
    }
    else
    {
        int atom_numbers = md_info->atom_numbers;
        excluded_atom_numbers = 0;
        controller->printf("    Set all atom exclude no atoms as default\n"); 

        Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
        Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);
        Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);

        Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
        Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
        Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);


        int count = 0;
        for (int i = 0; i < atom_numbers; i++)
        {
            h_excluded_numbers[i] = 0;
            h_excluded_list_start[i] = count;
            for (int j = 0; j < h_excluded_numbers[i]; j++)
            {
                h_excluded_list[count] = 0;
                count++;
            }
        }
        hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
    }

}

void MD_INFORMATION::periodic_box_condition_information::Initial(CONTROLLER *controller, MD_INFORMATION* md_info)
{
    this->md_info = md_info;
    this->uint_dr_to_dr_cof = 1.0f / CONSTANT_UINT_MAX_FLOAT * md_info->sys.box_length;
    this->md_info->MD_Information_Crd_To_Uint_Crd();
    this->pbc = true;
    if (controller->Command_Exist("pbc"))
    {
        this->pbc = controller->Get_Bool("pbc", "MD_INFORMATION::periodic_box_condition_information::Initial");
    }
    this->No_PBC_Check(controller);
}

void MD_INFORMATION::periodic_box_condition_information::No_PBC_Check(CONTROLLER* controller)
{
    if (this->pbc)
        return;
    if (md_info->nb.cutoff < 100)
    {
        controller->Warn("The cutoff for NOPBC is not greater than 100 angstrom, which may be inaccurate");
    }
    if (md_info->sys.box_length.x < 900 || md_info->sys.box_length.y < 900 || md_info->sys.box_length.z < 900)
    {
        controller->Throw_SPONGE_Error(spongeErrorConflictingCommand,
            "MD_INFORMATION::periodic_box_condition_information::No_PBC_Check",
            "The box length of the system should always be greater than 900 angstrom for NOPBC");
    }
    if (md_info->mode == md_info->NPT)
    {
        controller->Throw_SPONGE_Error(spongeErrorConflictingCommand,
            "MD_INFORMATION::periodic_box_condition_information::No_PBC_Check",
            "NPT mode can not be used for NOPBC");
    }
    if (controller->Command_Exist("SITS", "mode"))
    {
        controller->Throw_SPONGE_Error(spongeErrorConflictingCommand,
            "MD_INFORMATION::periodic_box_condition_information::No_PBC_Check",
            "SITS can not be used for NOPBC now");
    }
    Cuda_Malloc_Safely((void**)&nopbc_crd, sizeof(VECTOR) * (md_info->atom_numbers + md_info->no_direct_interaction_virtual_atom_numbers));
    hipMemset(nopbc_crd, 0, sizeof(VECTOR) * md_info->atom_numbers);
}

void MD_INFORMATION::Read_Mode(CONTROLLER *controller)
{
    if (controller->Command_Choice("mode", "nvt"))
    {
        controller->printf("    Mode set to NVT\n");
        mode = 1;
    }
    else if (controller->Command_Choice("mode", "npt"))
    {
        controller->printf("    Mode set to NPT\n");
        mode = 2;
    }
    else if (controller->Command_Choice("mode", "minimization"))
    {
        controller->printf("    Mode set to Energy Minimization\n");
        mode = -1;
    }
    else if (controller->Command_Choice("mode", "nve"))
    {
        controller->printf("    Mode set to NVE\n");
        mode = 0;
    }
    else if (controller->Command_Choice("mode", "rerun"))
    {
        controller->printf("    Mode set to RERUN\n");
        mode = -2;
    }
    else
    {
        controller->Throw_SPONGE_Error(spongeErrorMissingCommand, "MD_INFORMATION::Read_Mode", "Reason:\n\t'mode' is required\n");
    }
}

void MD_INFORMATION::Read_dt(CONTROLLER *controller)
{
    if (mode == RERUN)
    {
        return;
    }
    if (controller[0].Command_Exist("dt"))
    {
        controller->Check_Float("dt", "MD_INFORMATION::Read_dt");
        controller->printf("    dt set to %f ps\n", atof(controller[0].Command("dt")));
        dt = atof(controller[0].Command("dt")) * CONSTANT_TIME_CONVERTION;
        sscanf(controller[0].Command("dt"), "%lf", &sys.dt_in_ps);
    }
    else
    {
        if (mode != MINIMIZATION)
            dt = 0.001;
        else
            dt = 1e-8;
        sys.dt_in_ps = 0.001;
        controller->printf("    dt set to %e ps\n", dt);
        dt *= CONSTANT_TIME_CONVERTION;
    }
    if (mode == MINIMIZATION)
    {
        sys.dt_in_ps = 0;
    }
}

void MD_INFORMATION::trajectory_output::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    this->md_info = md_info;
    current_crd_synchronized_step = -1;
    if (md_info->mode != md_info->RERUN)
    {
        int default_interval = 1000;
        if (controller[0].Command_Exist("write_information_interval"))
        {
            controller->Check_Int("write_information_interval", "MD_INFORMATION::trajectory_output::Initial");
            default_interval = atoi(controller[0].Command("write_information_interval"));
        }
        write_trajectory_interval = default_interval;        
        if (controller[0].Command_Exist("write_trajectory_interval"))
        {
            controller->Check_Int("write_trajectory_interval", "MD_INFORMATION::trajectory_output::Initial");
            write_trajectory_interval = atoi(controller[0].Command("write_trajectory_interval"));
        }
        write_mdout_interval = default_interval;
        if (controller[0].Command_Exist("write_mdout_interval"))
        {
            controller->Check_Int("write_mdout_interval", "MD_INFORMATION::trajectory_output::Initial");
            write_mdout_interval = atoi(controller[0].Command("write_mdout_interval"));
        }
        write_restart_file_interval = md_info->sys.step_limit;
        if (controller[0].Command_Exist("write_restart_file_interval"))
        {
            controller->Check_Int("write_restart_file_interval", "MD_INFORMATION::trajectory_output::Initial");
            write_restart_file_interval = atoi(controller[0].Command("write_restart_file_interval"));
        }
        if (controller->Command_Exist(RESTART_COMMAND))
        {
            strcpy(restart_name, controller->Command(RESTART_COMMAND));
        }
        else
        {
            strcpy(restart_name, RESTART_DEFAULT_FILENAME);
        }
        if (controller->Command_Exist(FRC_TRAJ_COMMAND))
        {
            is_frc_traj = 1;
            Open_File_Safely(&frc_traj, controller->Command(FRC_TRAJ_COMMAND), "wb");
            controller->Set_File_Buffer(frc_traj, sizeof(VECTOR) * md_info->atom_numbers);
        }
        if (controller->Command_Exist(VEL_TRAJ_COMMAND))
        {
            is_vel_traj = 1;
            Open_File_Safely(&vel_traj, controller->Command(VEL_TRAJ_COMMAND), "wb");
            controller->Set_File_Buffer(vel_traj, sizeof(VECTOR) * md_info->atom_numbers);
        }
    }
    else
    {
        write_trajectory_interval = 0;
        write_mdout_interval = 1;
        write_restart_file_interval = 0;
    }

    if (write_trajectory_interval != 0)
    {
        if (controller->Command_Exist(TRAJ_COMMAND))
        {
            Open_File_Safely(&crd_traj, controller->Command(TRAJ_COMMAND), "wb");
        }
        else
        {
            Open_File_Safely(&crd_traj, TRAJ_DEFAULT_FILENAME, "wb");
        }
        controller->Set_File_Buffer(crd_traj, sizeof(VECTOR) * md_info->atom_numbers);
        if (controller->Command_Exist(BOX_TRAJ_COMMAND))
        {
            Open_File_Safely(&box_traj, controller->Command(BOX_TRAJ_COMMAND), "w");
        }
        else
        {
            Open_File_Safely(&box_traj, BOX_TRAJ_DEFAULT_FILENAME, "w");
        }
        controller->Set_File_Buffer(box_traj, sizeof(char) * 50);
    }
}

void MD_INFORMATION::NVE_iteration::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    this->md_info = md_info;
    max_velocity = -1;
    if (controller[0].Command_Exist("velocity_max"))
    {
        controller->Check_Float("velocity_max", "MD_INFORMATION::NVE_iteration::Initial");
        max_velocity = atof(controller[0].Command("velocity_max"));
    }
}
void MD_INFORMATION::residue_information::Read_AMBER_Parm7(const char *file_name, CONTROLLER controller)
{
    FILE *parm = NULL;
    Open_File_Safely(&parm, file_name, "r");
    controller.printf("    Start reading residue informataion from AMBER parm7:\n");
    
    while (true)
    {
        char temps[CHAR_LENGTH_MAX];
        char temp_first_str[CHAR_LENGTH_MAX];
        char temp_second_str[CHAR_LENGTH_MAX];
        if (fgets(temps, CHAR_LENGTH_MAX, parm) == NULL)
        {
            break;
        }
        if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
        {
            continue;
        }
        //read in atomnumber atomljtypenumber
        if (strcmp(temp_first_str, "%FLAG") == 0
            && strcmp(temp_second_str, "POINTERS") == 0)
        {
            char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);

            int atom_numbers = 0;
            int scanf_ret = fscanf(parm, "%d", &atom_numbers);
            if (scanf_ret != 1)
            {
                controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::residue_information::Read_AMBER_Parm7", "Reason:\n\tthe format of the amber_parm7 is not right\n");
            }
            if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
            {
                controller.Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::residue_information::Read_AMBER_Parm7", ATOM_NUMBERS_DISMATCH);
            }
            else if (md_info->atom_numbers == 0)
            {
                md_info->atom_numbers = atom_numbers;
            }
            for (int i = 0; i < 10; i = i + 1)
            {
                int lin;
                scanf_ret = fscanf(parm, "%d\n", &lin);
                if (scanf_ret != 1)
                {
                    controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::residue_information::Read_AMBER_Parm7", "Reason:\n\tthe format of the amber_parm7 is not right\n");
                }
            }
            scanf_ret = fscanf(parm, "%d\n", &this->residue_numbers);//NRES
            if (scanf_ret != 1)
            {
                controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::residue_information::Read_AMBER_Parm7", "Reason:\n\tthe format of the amber_parm7 is not right\n");
            }
            controller.printf("        residue_numbers is %d\n", this->residue_numbers);

            Malloc_Safely((void**)&h_mass, sizeof(float)*this->residue_numbers);
            Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*this->residue_numbers);
            Malloc_Safely((void**)&h_res_start, sizeof(int)*this->residue_numbers);
            Malloc_Safely((void**)&h_res_end, sizeof(int)*this->residue_numbers);
            Malloc_Safely((void**)&h_momentum, sizeof(float)*this->residue_numbers);
            Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
            Malloc_Safely((void**)&h_sigma_of_res_ek, sizeof(float));

            Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_res_start, sizeof(int)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_res_end, sizeof(int)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_momentum, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&res_ek_energy, sizeof(float)*this->residue_numbers); 
            Cuda_Malloc_Safely((void**)&sigma_of_res_ek, sizeof(float));
        }//FLAG POINTERS

        //residue range read
        if (strcmp(temp_first_str, "%FLAG") == 0
            && strcmp(temp_second_str, "RESIDUE_POINTER") == 0)
        {
            char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
            //注意读进来的数的编号要减1
            int *lin_serial;
            Malloc_Safely((void**)&lin_serial, sizeof(int)* this->residue_numbers);
            for (int i = 0; i<this->residue_numbers; i = i + 1)
            {
                int scanf_ret = fscanf(parm, "%d\n", &lin_serial[i]);
                if (scanf_ret != 1)
                {
                    controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::residue_information::Read_AMBER_Parm7", "Reason:\n\tthe format of the amber_parm7 is not right\n");
                }
            }
            for (int i = 0; i<this->residue_numbers - 1; i = i + 1)
            {
                h_res_start[i] = lin_serial[i] - 1;
                h_res_end[i] = lin_serial[i + 1] - 1;
            }
            h_res_start[this->residue_numbers - 1] = lin_serial[this->residue_numbers - 1] - 1;
            h_res_end[this->residue_numbers - 1] = md_info->atom_numbers + 1 - 1;

            free(lin_serial);
        }
    }//while cycle

    hipMemcpy(this->d_res_start, h_res_start, sizeof(int)*this->residue_numbers, hipMemcpyHostToDevice);
    hipMemcpy(this->d_res_end, h_res_end, sizeof(int)*this->residue_numbers, hipMemcpyHostToDevice);

    controller.printf("    End reading residue informataion from AMBER parm7\n\n");

    fclose(parm);
}

void MD_INFORMATION::residue_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    this->md_info = md_info;
    if (!(controller[0].Command_Exist("residue_in_file")))
    {
        if (controller[0].Command_Exist("amber_parm7"))
        {
            Read_AMBER_Parm7(controller[0].Command("amber_parm7"), controller[0]);
            is_initialized = 1;
        }
        //对于没有residue输入的模拟，默认每个粒子作为一个residue
        else
        {
            residue_numbers = md_info->atom_numbers;
            controller->printf("    Set default residue list:\n");
            controller->printf("        residue_numbers is %d\n", residue_numbers);
            Malloc_Safely((void**)&h_mass, sizeof(float)*this->residue_numbers);
            Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*this->residue_numbers);
            Malloc_Safely((void**)&h_res_start, sizeof(int)*this->residue_numbers);
            Malloc_Safely((void**)&h_res_end, sizeof(int)*this->residue_numbers);
            Malloc_Safely((void**)&h_momentum, sizeof(float)*this->residue_numbers);
            Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
            Malloc_Safely((void**)&h_sigma_of_res_ek, sizeof(float));

            Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_res_start, sizeof(int)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_res_end, sizeof(int)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_momentum, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&res_ek_energy, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&sigma_of_res_ek, sizeof(float));
            int count = 0;
            int temp=1;//每个粒子作为一个residue
            for (int i = 0; i < residue_numbers; i++)
            {
                h_res_start[i] = count;
                count += temp;
                h_res_end[i] = count;
            }
            hipMemcpy(d_res_start, h_res_start, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
            hipMemcpy(d_res_end, h_res_end, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
            controller->printf("    End reading residue list\n\n");
            is_initialized = 1;
        }
    }
    else
    {
        FILE *fp = NULL;
        controller->printf("    Start reading residue list:\n");
        Open_File_Safely(&fp, controller[0].Command("residue_in_file"), "r");
        int atom_numbers = 0;
        int scanf_ret = fscanf(fp, "%d %d", &atom_numbers, &residue_numbers);
        if (scanf_ret != 2)
        {
            controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::residue_information::Initial", "Reason:\n\tthe format of the residue_in_file is not right\n");
        }
        if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
        {
            controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::residue_information::Initial", ATOM_NUMBERS_DISMATCH);
        }
        else if (md_info->atom_numbers == 0)
        {
            md_info->atom_numbers = atom_numbers;
        }
        controller->printf("        residue_numbers is %d\n", residue_numbers);
        Malloc_Safely((void**)&h_mass, sizeof(float)*this->residue_numbers);
        Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*this->residue_numbers);
        Malloc_Safely((void**)&h_res_start, sizeof(int)*this->residue_numbers);
        Malloc_Safely((void**)&h_res_end, sizeof(int)*this->residue_numbers);
        Malloc_Safely((void**)&h_momentum, sizeof(float)*this->residue_numbers);
        Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
        Malloc_Safely((void**)&h_sigma_of_res_ek, sizeof(float));

        Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&d_res_start, sizeof(int)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&d_res_end, sizeof(int)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&d_momentum, sizeof(float)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&res_ek_energy, sizeof(float)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&sigma_of_res_ek, sizeof(float));

        int count = 0;
        int temp;
        for (int i = 0; i < residue_numbers; i++)
        {
            h_res_start[i] = count;
            scanf_ret = fscanf(fp, "%d", &temp);
            if (scanf_ret != 1)
            {
                controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::residue_information::Initial", "Reason:\n\tthe format of the residue_in_file is not right\n");
            }
            count += temp;
            h_res_end[i] = count;
        }
        hipMemcpy(d_res_start, h_res_start, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_res_end, h_res_end, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
        controller->printf("    End reading residue list\n\n");
        fclose(fp);
        is_initialized = 1;
    }
    if (is_initialized)
    {
        if (md_info->h_mass != NULL)
        {
            for (int i = 0; i < residue_numbers; i++)
            {
                float temp_mass = 0;
                for (int j = h_res_start[i]; j < h_res_end[i]; j++)
                {
                    temp_mass += md_info->h_mass[j];
                }
                this->h_mass[i] = temp_mass;
                if (temp_mass == 0)
                    this->h_mass_inverse[i] = 0;
                else
                    this->h_mass_inverse[i] = 1.0 / temp_mass;
            }
            hipMemcpy(d_mass_inverse, h_mass_inverse, sizeof(float)* residue_numbers, hipMemcpyHostToDevice);
            hipMemcpy(d_mass, h_mass, sizeof(float)* residue_numbers, hipMemcpyHostToDevice);
        }
        else
        {
            controller->Throw_SPONGE_Error(spongeErrorMissingCommand,
                                           "MD_INFORMATION::residue_information::Initial",
                                           "Reason:\n\tno mass information found");
        }
    }
}

void MD_INFORMATION::Read_Coordinate_And_Velocity(CONTROLLER *controller)
{
    sys.start_time = 0.0;
    if (mode == RERUN)
    {
        if (atom_numbers == 0)
        {
            controller->Throw_SPONGE_Error(spongeErrorMissingCommand, "MD_INFORMATION::Read_Coordinate_And_Velocity", 
                "Reason:\n\tFor the 'rerun' mode, the number of atoms should be provided by mass_in_file or charge_in_file\n");
        }
        this->atom_numbers = atom_numbers;
        Malloc_Safely((void**)&coordinate, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
        Cuda_Malloc_Safely((void**)&crd, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));   
        Cuda_Malloc_Safely((void**)&last_crd, sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
        hipMemset(last_crd, 0,  sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));         
        Malloc_Safely((void**)&velocity, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
        Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
        sys.box_length = { 100.0f, 100.0f, 100.0f };
        rerun.Initial(controller, this);
        rerun.Iteration(rerun.start_frame);
        sys.box_length.x *= rerun.box_length_change_factor.x;
        sys.box_length.y *= rerun.box_length_change_factor.y;
        sys.box_length.z *= rerun.box_length_change_factor.z;
    }
    else if (controller[0].Command_Exist("coordinate_in_file"))
    {
        Read_Coordinate_In_File(controller[0].Command("coordinate_in_file"), controller[0]);
        if (controller[0].Command_Exist("velocity_in_file"))
        {
            FILE *fp = NULL;
            controller->printf("    Start reading velocity_in_file:\n");
            Open_File_Safely(&fp, controller[0].Command("velocity_in_file"), "r");
            
            int atom_numbers = 0;
            char lin[CHAR_LENGTH_MAX];
            char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fp);
            int scanf_ret = sscanf(lin, "%d", &atom_numbers);
            if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
            {
                controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::Read_Coordinate_And_Velocity", ATOM_NUMBERS_DISMATCH);
            }
            Malloc_Safely((void**)&velocity, sizeof(VECTOR)* (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
            Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR)* (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
            for (int i = 0; i < atom_numbers; i++)
            {
                scanf_ret = fscanf(fp, "%f %f %f", &velocity[i].x, &velocity[i].y, &velocity[i].z);
                if (scanf_ret != 3)
                {
                    std::string error_reason = "Reason:\n\tthe format of the velocity_in_file (";
                    error_reason += controller->Command("velocity_in_file");
                    error_reason += ") is not right (missing the velocity of atom ";
                    error_reason += i;
                    error_reason += ")\n";
                    controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Coordinate_And_Velocity", error_reason.c_str());
                }
            }
            hipMemcpy(vel, velocity, sizeof(VECTOR)* atom_numbers, hipMemcpyHostToDevice);
            controller->printf("    End reading velocity_in_file\n\n");
            fclose(fp);
        }
        else
        {
            controller->printf("    Velocity is set to zero as default\n");
            Malloc_Safely((void**)&velocity, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
            Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
            for (int i = 0; i < atom_numbers; i++)
            {
                velocity[i].x = 0;
                velocity[i].y = 0;
                velocity[i].z = 0;
            }
            hipMemcpy(vel, velocity, sizeof(VECTOR)* atom_numbers, hipMemcpyHostToDevice);
        }
    }
    else if (controller[0].Command_Exist("amber_rst7"))
    {
        output.amber_irest = 1;
        if (controller[0].Command_Exist("amber_irest"))
        {
            output.amber_irest = controller->Get_Bool("amber_irest", "MD_INFORMATION::Read_Coordinate_And_Velocity");
        }
        Read_Rst7(controller[0].Command("amber_rst7"), output.amber_irest, controller[0]);
    }
    else
    {
        controller->Throw_SPONGE_Error(spongeErrorMissingCommand,
            "MD_INFORMATION::Read_Coordinate_And_Velocity",
            "Reason:\n\tno coordinate information found");
    }
}

void MD_INFORMATION::Read_Mass(CONTROLLER *controller)
{
    if (controller[0].Command_Exist("mass_in_file"))
    {
        FILE *fp = NULL;
        controller->printf("    Start reading mass:\n");
        Open_File_Safely(&fp, controller[0].Command("mass_in_file"), "r");
        int atom_numbers = 0;
        char lin[CHAR_LENGTH_MAX];
        char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fp);
        int scanf_ret = sscanf(lin, "%d", &atom_numbers);
        if (scanf_ret != 1)
        {
            controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of mass_in_file is not right\n");
        }
        if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
        {
            controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::Read_Mass", ATOM_NUMBERS_DISMATCH);
        }
        else if (this->atom_numbers == 0)
        {
            this->atom_numbers = atom_numbers;
        }
        Malloc_Safely((void**)&h_mass, sizeof(float)* atom_numbers);
        Malloc_Safely((void**)&h_mass_inverse, sizeof(float)* atom_numbers);
        Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)* atom_numbers);
        Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)* atom_numbers);
        sys.total_mass = 0;
        for (int i = 0; i < atom_numbers; i++)
        {
            scanf_ret = fscanf(fp, "%f", &h_mass[i]);
            if (scanf_ret != 1)
            {
                controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of mass_in_file is not right\n");
            }
            sys.total_mass += h_mass[i];
            if (h_mass[i] == 0)
                h_mass_inverse[i] = 0;
            else
                h_mass_inverse[i] = 1.0 / h_mass[i];

        }
        controller->printf("    End reading mass\n\n");
        fclose(fp);
    }
    else if (controller[0].Command_Exist("amber_parm7"))
    {
        FILE *parm = NULL;
        Open_File_Safely(&parm, controller[0].Command("amber_parm7"), "r");
        controller[0].printf("    Start reading mass from AMBER parm7:\n");
        while (true)
        {
            char temps[CHAR_LENGTH_MAX];
            char temp_first_str[CHAR_LENGTH_MAX];
            char temp_second_str[CHAR_LENGTH_MAX];
            if (fgets(temps, CHAR_LENGTH_MAX, parm) == NULL)
            {
                break;
            }
            if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
            {
                continue;
            }
            //read in atomnumber atomljtypenumber
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "POINTERS") == 0)
            {
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);

                int atom_numbers = 0;
                int scanf_ret = fscanf(parm, "%d", &atom_numbers);
                if (scanf_ret != 1)
                {
                    controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                }
                if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
                {
                    controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::Read_Mass", ATOM_NUMBERS_DISMATCH);
                }
                else if (this->atom_numbers == 0)
                {
                    this->atom_numbers = atom_numbers;
                }
                Malloc_Safely((void**)&h_mass, sizeof(float)* atom_numbers);
                Malloc_Safely((void**)&h_mass_inverse, sizeof(float)* atom_numbers);
                Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)* atom_numbers);
                Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)* atom_numbers);
            }
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "MASS") == 0)
            {
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
                double lin;
                sys.total_mass = 0;
                for (int i = 0; i < this->atom_numbers; i = i + 1)
                {
                    int scanf_ret = fscanf(parm, "%lf\n", &lin);
                    if (scanf_ret != 1)
                    {
                        controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                    }
                    this->h_mass[i] = (float)lin;
                    if (h_mass[i] == 0)
                        h_mass_inverse[i] = 0;
                    else
                        h_mass_inverse[i] = 1.0f / h_mass[i];
                    sys.total_mass += h_mass[i];
                }
            }
        }
        controller[0].printf("    End reading mass from AMBER parm7\n\n");
        fclose(parm);
    }
    else if (atom_numbers > 0)
    {
        controller[0].printf("    mass is set to 20 as default\n");
        sys.total_mass = 0;
        Malloc_Safely((void**)&h_mass, sizeof(float)* atom_numbers);
        Malloc_Safely((void**)&h_mass_inverse, sizeof(float)* atom_numbers);
        Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)* atom_numbers);
        Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)* atom_numbers);
        for (int i = 0; i < atom_numbers; i++)
        {
            h_mass[i] = 20;
            h_mass_inverse[i] = 1.0 / h_mass[i];
            sys.total_mass += h_mass[i];
        }
    }
    else
    {
        controller->Throw_SPONGE_Error(spongeErrorMissingCommand, "MD_INFORMATION::Read_Mass", ATOM_NUMBERS_MISSING);
    }
    if (atom_numbers > 0)
    {
        hipMemcpy(d_mass, h_mass, sizeof(float)* atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_mass_inverse, h_mass_inverse, sizeof(float)* atom_numbers, hipMemcpyHostToDevice);
    }
}

void MD_INFORMATION::Read_Charge(CONTROLLER *controller)
{
    if (controller[0].Command_Exist("charge_in_file"))
    {
        FILE *fp = NULL;
        controller->printf("    Start reading charge:\n");
        Open_File_Safely(&fp, controller[0].Command("charge_in_file"), "r");
        int atom_numbers = 0;
        char lin[CHAR_LENGTH_MAX];
        char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fp);
        int scanf_ret = sscanf(lin, "%d", &atom_numbers);
        if (scanf_ret != 1)
        {
            controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of charge_in_file is not right\n");
        }
        if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
        {
            controller->Throw_SPONGE_Error(spongeErrorMissingCommand, "MD_INFORMATION::Read_Charge", ATOM_NUMBERS_DISMATCH);
        }
        else if (this->atom_numbers == 0)
        {
            this->atom_numbers = atom_numbers;
        }
        Malloc_Safely((void**)&h_charge, sizeof(float)* atom_numbers);
        Cuda_Malloc_Safely((void**)&d_charge, sizeof(float)* atom_numbers);
        for (int i = 0; i < atom_numbers; i++)
        {
            scanf_ret = fscanf(fp, "%f", &h_charge[i]);
            if (scanf_ret != 1)
            {
                controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of charge_in_file is not right\n");
            }
        }
        controller->printf("    End reading charge\n\n");
        fclose(fp);
    }
    else if (controller[0].Command_Exist("amber_parm7"))
    {
        FILE *parm = NULL;
        Open_File_Safely(&parm, controller[0].Command("amber_parm7"), "r");
        controller[0].printf("    Start reading charge from AMBER parm7:\n");
        while (true)
        {
            char temps[CHAR_LENGTH_MAX];
            char temp_first_str[CHAR_LENGTH_MAX];
            char temp_second_str[CHAR_LENGTH_MAX];
            if (fgets(temps, CHAR_LENGTH_MAX, parm) == NULL)
            {
                break;
            }
            if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
            {
                continue;
            }
            //read in atomnumber atomljtypenumber
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "POINTERS") == 0)
            {
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);

                int atom_numbers = 0;
                int scanf_ret = fscanf(parm, "%d", &atom_numbers);
                if (scanf_ret != 1)
                {
                    controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                }
                if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
                {
                    controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::Read_Charge", ATOM_NUMBERS_DISMATCH);
                }
                else if (this->atom_numbers == 0)
                {
                    this->atom_numbers = atom_numbers;
                }
                Malloc_Safely((void**)&h_charge, sizeof(float)* atom_numbers);
                Cuda_Malloc_Safely((void**)&d_charge, sizeof(float)* atom_numbers);
            }
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "CHARGE") == 0)
            {
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
                for (int i = 0; i < this->atom_numbers; i = i + 1)
                {
                    int scanf_ret = fscanf(parm, "%f", &h_charge[i]);
                    if (scanf_ret != 1)
                    {
                        controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                    }
                }
            }
        }
        controller[0].printf("    End reading charge from AMBER parm7\n\n");
        fclose(parm);
    }
    else if (atom_numbers > 0)
    {
        controller[0].printf("    charge is set to 0 as default\n");
        Malloc_Safely((void**)&h_charge, sizeof(float)* atom_numbers);
        Cuda_Malloc_Safely((void**)&d_charge, sizeof(float)* atom_numbers);
        for (int i = 0; i < atom_numbers; i++)
        {
            h_charge[i] = 0;
        }
    }
    else
    {
        controller->Throw_SPONGE_Error(spongeErrorMissingCommand, "MD_INFORMATION::Read_Charge", ATOM_NUMBERS_MISSING);
    }
    if (atom_numbers > 0)
    {
        hipMemcpy(d_charge, h_charge, sizeof(float)* atom_numbers, hipMemcpyHostToDevice);
    }
}

//MD_INFORMATION成员函数
void MD_INFORMATION::Initial(CONTROLLER *controller)
{
    controller->printf("START INITIALIZING MD CORE:\n");
    atom_numbers = 0; //初始化，使得能够进行所有原子数目是否相等的判断
    
    strcpy(md_name, controller[0].Command("md_name"));
    Read_Mode(controller);
    Read_dt(controller);

    Read_Mass(controller);
    Read_Charge(controller);
    Atom_Information_Initial();

    Read_Coordinate_And_Velocity(controller);

    sys.Initial(controller, this);
    nb.Initial(controller, this);
    
    output.Initial(controller, this);

    nve.Initial(controller, this);
    
    min.Initial(controller, this);

    res.Initial(controller, this);

    mol.md_info = this;

    pbc.Initial(controller, this);
    
    is_initialized = 1;
    controller->printf("    structure last modify date is %d\n", last_modify_date);
    controller->printf("END INITIALIZING MD CORE\n\n");
}

void MD_INFORMATION::Atom_Information_Initial()
{
    Malloc_Safely((void**)&this->force, sizeof(VECTOR) * (atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Malloc_Safely((void**)&this->h_atom_energy, sizeof(float) * atom_numbers);
    Malloc_Safely((void**)&this->h_atom_virial, sizeof(double)* atom_numbers);
    Cuda_Malloc_Safely((void**)&this->acc, sizeof(VECTOR) * atom_numbers);
    Cuda_Malloc_Safely((void**)&this->frc, sizeof(VECTOR) * (atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Cuda_Malloc_Safely((void**)&this->uint_crd, sizeof(UNSIGNED_INT_VECTOR) * (atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Cuda_Malloc_Safely((void**)&this->d_atom_energy, sizeof(float) * atom_numbers);
    Cuda_Malloc_Safely((void**)&this->d_atom_virial, sizeof(float) * atom_numbers);
    Cuda_Malloc_Safely((void**)&this->d_atom_ek, sizeof(float) * atom_numbers);
    Reset_List << <ceilf((float)3.*this->atom_numbers / 32), 32 >> >
        (3 * this->atom_numbers, (float*)this->acc, 0.);
    Reset_List << <ceilf((float)3.*this->atom_numbers / 32), 32 >> >
        (3 * this->atom_numbers, (float*)this->frc, 0.);
    sys.freedom = 3 * atom_numbers; //最大自由度，后面减
}

void MD_INFORMATION::Read_Coordinate_In_File(const char* file_name, CONTROLLER controller)
{
    FILE *fp =NULL;
    controller.printf("    Start reading coordinate_in_file:\n");
    Open_File_Safely(&fp, file_name, "r");
    char lin[CHAR_LENGTH_MAX];
    char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fp);
    int atom_numbers = 0;
    int scanf_ret = sscanf(lin, "%d %lf", &atom_numbers, &sys.start_time);
    if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
    {
        controller.Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::Read_Coordinate_In_File", ATOM_NUMBERS_DISMATCH);
    }
    else if (this->atom_numbers == 0)
    {
        this->atom_numbers = atom_numbers;
    }
    if (scanf_ret == 0)
    {
        std::string error_reason = "Reason:\n\tthe format of the coordinate_in_file (";
        error_reason += file_name;
        error_reason += ") is not right\n";
        controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Coordinate_In_File", error_reason.c_str());
    }
    else  if (scanf_ret == 1)
    {
        sys.start_time = 0;
    }

    controller.printf("        atom_numbers is %d\n", this->atom_numbers);
    controller.printf("        system start_time is %lf\n", this->sys.start_time);
    Malloc_Safely((void**)&coordinate, sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Cuda_Malloc_Safely((void**)&crd, sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Cuda_Malloc_Safely((void**)&last_crd, sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
    hipMemset(last_crd, 0,  sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));

    for (int i = 0; i < atom_numbers; i++)
    {
        scanf_ret = fscanf(fp, "%f %f %f", &coordinate[i].x, &coordinate[i].y, &coordinate[i].z);
        if (scanf_ret != 3)
        {
            std::string error_reason = "Reason:\n\tthe format of the coordinate_in_file (";
            error_reason += file_name;
            error_reason += ") is not right\n";
            controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Coordinate_In_File", error_reason.c_str());
        }
    }
    scanf_ret = fscanf(fp, "%f %f %f", &sys.box_length.x, &sys.box_length.y, &sys.box_length.z);
    if (scanf_ret != 3)
    {
        std::string error_reason = "Reason:\n\tthe format of the coordinate_in_file (";
        error_reason += file_name;
        error_reason += ") is not right\n";
        controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Coordinate_In_File", error_reason.c_str());
    }
    controller.printf("        box_length is\n            x: %f\n            y: %f\n            z: %f\n", sys.box_length.x, sys.box_length.y, sys.box_length.z);
    hipMemcpy(crd, coordinate, sizeof(VECTOR)* atom_numbers, hipMemcpyHostToDevice);
    controller.printf("    End reading coordinate_in_file\n\n");
    fclose(fp);
}
void MD_INFORMATION::Read_Rst7(const char* file_name, int irest, CONTROLLER controller)
{
    FILE *fin = NULL;
    Open_File_Safely(&fin, file_name, "r");
    controller.printf("    Start reading AMBER rst7:\n");
    char lin[CHAR_LENGTH_MAX];
    int atom_numbers = 0;
    char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fin);
    get_ret = fgets(lin, CHAR_LENGTH_MAX, fin);
    int has_vel = 0;
    int scanf_ret = sscanf(lin, "%d %lf", &atom_numbers, &sys.start_time);
    if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
    {
        controller.Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::Read_Rst7", ATOM_NUMBERS_DISMATCH);
    }
    else if (this->atom_numbers == 0)
    {
        this->atom_numbers = atom_numbers;
    }
    if (scanf_ret == 0)
    {
        std::string error_reason = "Reason:\n\tthe format of the amber_rst7 (";
        error_reason += file_name;
        error_reason += ") is not right\n";
        controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Rst7", error_reason.c_str());
    }
    else  if (scanf_ret == 2)
    {
        has_vel = 1;
    }
    else
    {
        sys.start_time = 0;
    }

    Malloc_Safely((void**)&coordinate, sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Cuda_Malloc_Safely((void**)&crd, sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Cuda_Malloc_Safely((void**)&last_crd, sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
    hipMemset(last_crd, 0,  sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Malloc_Safely((void**)&velocity, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));

    controller.printf("        atom_numbers is %d\n", this->atom_numbers);
    controller.printf("        system start time is %lf\n", this->sys.start_time);

    if (has_vel == 0 || irest == 0)
    {
        controller.printf("        All velocity will be set to 0\n");
    }


    for (int i = 0; i < this->atom_numbers; i = i + 1)
    {
        scanf_ret = fscanf(fin, "%f %f %f",
            &this->coordinate[i].x,
            &this->coordinate[i].y,
            &this->coordinate[i].z);
        if (scanf_ret != 3)
        {
            std::string error_reason = "Reason:\n\tthe format of the amber_rst7 (";
            error_reason += file_name;
            error_reason += ") is not right (missing the coordinate of atom ";
            error_reason += i;
            error_reason += ")\n";
            controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Rst7", error_reason.c_str());
        }
    }
    if (has_vel)
    {
        for (int i = 0; i < this->atom_numbers; i = i + 1)
        {
            scanf_ret = fscanf(fin, "%f %f %f",
                &this->velocity[i].x,
                &this->velocity[i].y,
                &this->velocity[i].z);
            if (scanf_ret != 3)
            {
                std::string error_reason = "Reason:\n\tthe format of the amber_rst7 (";
                error_reason += file_name;
                error_reason += ") is not right (missing the velocity of atom ";
                error_reason += i;
                error_reason += ")\n";
                controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Rst7", error_reason.c_str());
            }
        }
    }
    if (irest == 0 || !has_vel)
    {
        for (int i = 0; i < this->atom_numbers; i = i + 1)
        {
            this->velocity[i].x = 0.0;
            this->velocity[i].y = 0.0;
            this->velocity[i].z = 0.0;
        }
    }
    scanf_ret = fscanf(fin, "%f %f %f", &this->sys.box_length.x, &this->sys.box_length.y, &this->sys.box_length.z);
    if (scanf_ret != 3)
    {
        std::string error_reason = "Reason:\n\tthe format of the amber_rst7 (";
        error_reason += file_name;
        error_reason += ") is not right (missing the box information)\n";
        controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Rst7", error_reason.c_str());
    }
    controller.printf("        system size is %f %f %f\n", this->sys.box_length.x, this->sys.box_length.y, this->sys.box_length.z);
    hipMemcpy(this->crd, this->coordinate, sizeof(VECTOR)*this->atom_numbers, hipMemcpyHostToDevice);
    hipMemcpy(this->vel, this->velocity, sizeof(VECTOR)*this->atom_numbers, hipMemcpyHostToDevice);
    fclose(fin);
    controller.printf("    End reading AMBER rst7\n\n");
}

void MD_INFORMATION::trajectory_output::Append_Crd_Traj_File(FILE *fp)
{
    if (md_info->is_initialized)
    {
        md_info->Crd_Vel_Device_To_Host();
        if (fp == NULL)
        {
            fp = crd_traj;
        }
        fwrite(&md_info->coordinate[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
    }
}

// 20210827用于输出速度和力
void MD_INFORMATION::trajectory_output::Append_Frc_Traj_File(FILE *fp)
{
    if (md_info->is_initialized)
    {
        hipMemcpy(md_info->force, md_info->frc, sizeof(VECTOR)*md_info->atom_numbers, hipMemcpyDeviceToHost);
        if (fp == NULL)//默认的frc输出位置
        {
            fp = frc_traj;
            if (fp != NULL)
            {
                fwrite(&md_info->force[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
            }
        }
        else
        {
            fwrite(&md_info->force[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
        }
    }
}
void MD_INFORMATION::trajectory_output::Append_Vel_Traj_File(FILE *fp)
{
    if (md_info->is_initialized)
    {
        hipMemcpy(md_info->velocity, md_info->vel, sizeof(VECTOR)*md_info->atom_numbers, hipMemcpyDeviceToHost);
        if (fp == NULL)//默认的vel输出位置
        {
            fp = vel_traj;
            if (fp != NULL)
            {
                fwrite(&md_info->velocity[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
            }
        }
        else
        {
            fwrite(&md_info->velocity[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
        }
    }
}

void MD_INFORMATION::trajectory_output::Append_Box_Traj_File(FILE *fp)
{
    if (md_info->is_initialized)
    {
        if (fp == NULL)
        {
            fp = box_traj;
        }
        fprintf(fp, "%f %f %f %.0f %.0f %.0f\n", md_info->sys.box_length.x, md_info->sys.box_length.y, md_info->sys.box_length.z, 90.0f, 90.0f, 90.0f);
    }
}

void MD_INFORMATION::trajectory_output::Export_Restart_File(const char *rst7_name)
{
    if (!md_info->is_initialized)
        return;

    char filename[CHAR_LENGTH_MAX];
    if (rst7_name == NULL)
        strcpy(filename, restart_name);
    else
        strcpy(filename, rst7_name);
    md_info->Crd_Vel_Device_To_Host();
    if (amber_irest >= 0)
    {
        const char *sys_name = md_info->md_name;
        FILE *lin = NULL;
        Open_File_Safely(&lin, filename, "w");
        fprintf(lin, "%s\n", sys_name);
        fprintf(lin, "%8d %.3lf\n", md_info->atom_numbers, md_info->sys.Get_Current_Time());
        int s = 0;
        for (int i = 0; i < md_info->atom_numbers; i = i + 1)
        {
            fprintf(lin, "%12.7f%12.7f%12.7f", md_info->coordinate[i].x, md_info->coordinate[i].y, md_info->coordinate[i].z);
            s = s + 1;
            if (s == 2)
            {
                s = 0;
                fprintf(lin, "\n");
            }
        }
        if (s == 1)
        {
            s = 0;
            fprintf(lin, "\n");
        }
        for (int i = 0; i < md_info->atom_numbers; i = i + 1)
        {
            fprintf(lin, "%12.7f%12.7f%12.7f", md_info->velocity[i].x, md_info->velocity[i].y, md_info->velocity[i].z);
            s = s + 1;
            if (s == 2)
            {
                s = 0;
                fprintf(lin, "\n");
            }
        }
        if (s == 1)
        {
            s = 0;
            fprintf(lin, "\n");
        }
        fprintf(lin, "%12.7f%12.7f%12.7f", (float)md_info->sys.box_length.x, (float)md_info->sys.box_length.y, (float)md_info->sys.box_length.z);
        fprintf(lin, "%12.7f%12.7f%12.7f", (float)90., (float)90., (float)90.);
        fclose(lin);
    }
    else
    {
        FILE *lin = NULL;
        FILE *lin2 = NULL;
        std::string buffer;
        buffer = std::string(filename) + "_coordinate.txt";
        Open_File_Safely(&lin, buffer.c_str(), "w");
        buffer = std::string(filename) + "_velocity.txt";
        Open_File_Safely(&lin2, buffer.c_str(), "w");
        fprintf(lin, "%d %.3lf\n", md_info->atom_numbers, md_info->sys.Get_Current_Time());
        fprintf(lin2, "%d %.3lf\n", md_info->atom_numbers, md_info->sys.Get_Current_Time());
        for (int i = 0; i < md_info->atom_numbers; i++)
        {
            fprintf(lin, "%12.7f %12.7f %12.7f\n", md_info->coordinate[i].x, md_info->coordinate[i].y, md_info->coordinate[i].z);
            fprintf(lin2, "%12.7f %12.7f %12.7f\n", md_info->velocity[i].x, md_info->velocity[i].y, md_info->velocity[i].z);
        }
        fprintf(lin, "%12.7f %12.7f %12.7f %12.7f %12.7f %12.7f", md_info->sys.box_length.x, md_info->sys.box_length.y, md_info->sys.box_length.z, 90.0f, 90.0f, 90.0f);
        fclose(lin);
        fclose(lin2);
    }
}


void MD_INFORMATION::Update_Volume(double factor)
{
    sys.box_length = factor * sys.box_length;
    pbc.uint_dr_to_dr_cof = 1.0f / CONSTANT_UINT_MAX_FLOAT * sys.box_length;
    MD_Information_Crd_To_Uint_Crd();
}

void MD_INFORMATION::Update_Box_Length(VECTOR factor)
{
    sys.box_length.x = factor.x * sys.box_length.x;
    sys.box_length.y = factor.y * sys.box_length.y;
    sys.box_length.z = factor.z * sys.box_length.z;
    pbc.uint_dr_to_dr_cof = 1.0f / CONSTANT_UINT_MAX_FLOAT * sys.box_length;
    MD_Information_Crd_To_Uint_Crd();
}

float MD_INFORMATION::system_information::Get_Density()
{
    density = total_mass * 1e24f / 6.023e23f / Get_Volume();
    return density;
}

double MD_INFORMATION::system_information::Get_Current_Time()
{
    current_time = start_time + (double)dt_in_ps * steps;
    return current_time;
}

float MD_INFORMATION::system_information::Get_Volume()
{
    volume = box_length.x * box_length.y * box_length.z;
    return volume;
}

void MD_INFORMATION::MD_Information_Crd_To_Uint_Crd()
{
    Crd_To_Uint_Crd << <(atom_numbers + no_direct_interaction_virtual_atom_numbers + 127) / 128, 128 >> >
        (this->atom_numbers + no_direct_interaction_virtual_atom_numbers, 1.0f / sys.box_length, crd, uint_crd);
}

void MD_INFORMATION::NVE_iteration::Leap_Frog()
{
    if (max_velocity <= 0)
    {
        MD_Iteration_Leap_Frog << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
            (md_info->atom_numbers, md_info->vel, md_info->crd, md_info->frc, md_info->acc, md_info->d_mass_inverse, md_info->dt);
    }
    else
    {
        MD_Iteration_Leap_Frog_With_Max_Velocity << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
            (md_info->atom_numbers, md_info->vel, md_info->crd, md_info->frc, md_info->acc, md_info->d_mass_inverse, md_info->dt, max_velocity);
    }
}

void MD_INFORMATION::MINIMIZATION_iteration::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    this->md_info = md_info;
    if (md_info->mode == MINIMIZATION)
    {
        controller->printf("    Start initializing minimization:\n");
        max_move = 0.1f;
        if (controller[0].Command_Exist("minimization_max_move"))
        {
            controller->Check_Float("minimization", "max_move", "MD_INFORMATION::MINIMIZATION_iteration::Initial");
            max_move = atof(controller[0].Command("minimization_max_move"));
        }
        controller->printf("        minimization max move is %f A\n", max_move);

        momentum_keep = 0;
        if (controller[0].Command_Exist("minimization_momentum_keep"))
        {
            controller->Check_Float("minimization", "momentum_keep", "MD_INFORMATION::MINIMIZATION_iteration::Initial");
            momentum_keep = atof(controller[0].Command("minimization_momentum_keep"));
        }
        controller->printf("        minimization momentum keep is %f\n", momentum_keep);

        dynamic_dt = 1;
        if (controller[0].Command_Exist("minimization_dynamic_dt"))
        {
            controller->Check_Int("minimization", "dynamic_dt", "MD_INFORMATION::MINIMIZATION_iteration::Initial");
            dynamic_dt = atoi(controller[0].Command("minimization_dynamic_dt"));
        }
        controller->printf("        minimization dynamic dt is %d\n", dynamic_dt);

        if (dynamic_dt)
        {
            dt_decreasing_rate = 0.01;
            if (controller[0].Command_Exist("minimization_dt_decreasing_rate"))
            {
                controller->Check_Float("minimization", "dt_decreasing_rate", "MD_INFORMATION::MINIMIZATION_iteration::Initial");
                dt_decreasing_rate = atof(controller[0].Command("minimization_dt_decreasing_rate"));
            }
            controller->printf("        minimization dt decreasing rate is %f\n", dt_decreasing_rate);

            dt_increasing_rate = 1.01 + rand() / RAND_MAX;
            if (controller[0].Command_Exist("minimization_dt_increasing_rate"))
            {
                controller->Check_Float("minimization", "dt_increasing_rate", "MD_INFORMATION::MINIMIZATION_iteration::Initial");
                dt_increasing_rate = atof(controller[0].Command("minimization_dt_increasing_rate"));
            }
            controller->printf("        minimization dt increasing rate is %f\n", dt_increasing_rate);
    
           dt_factor = 1e-4 + 1e-2 * rand() / RAND_MAX;
            if (controller[0].Command_Exist("minimization_dt_factor"))
            {
                controller->Check_Float("minimization", "dt_factor", "MD_INFORMATION::MINIMIZATION_iteration::Initial");
                dt_factor = atof(controller[0].Command("minimization_dt_factor"));
            }
            controller->printf("        minimization dt factor is %f\n", dt_factor);
            last_decrease_step = 0;
            Cuda_Malloc_Safely((void**)&d_max_force, sizeof(float));
        }
        controller->printf("    End initializing minimization\n\n");
    }
}

static __global__ void find_max_frc(int atom_numbers,  VECTOR *frc,  float *d_max_force)
{
    __shared__ float maxf[1024];
    float temp_f, temp_max = 0;
    VECTOR temp_frc;
    for (int i = threadIdx.x; i < atom_numbers; i += blockDim.x)
    {
        temp_frc = frc[i];
        temp_f = norm3df(temp_frc.x, temp_frc.y, temp_frc.z);
        if (temp_f > temp_max)
        {
            temp_max = temp_f;
        }
    }
    maxf[threadIdx.x] = temp_max;
    __syncthreads();
    int delta = 1024;
    for (int i = 0; i < 10; i += 1)
    {
        delta >>= 1;
        if (threadIdx.x < delta)
        {
            maxf[threadIdx.x] = fmaxf(maxf[threadIdx.x], maxf[threadIdx.x + delta]);
        }
    }
    d_max_force[0] = maxf[0];
}

void MD_INFORMATION::MINIMIZATION_iteration::Gradient_Descent()
{
    if (dynamic_dt)
    {
        if (md_info->sys.steps != 1)
        {
            if (last_decrease_step == 0)
            {
                if (last_potential > md_info->sys.h_potential)
                {
                    md_info->dt *= dt_increasing_rate;
                }
                else
                {
                    if (md_info->dt > 1e-8f)
                    {
                        md_info->dt *= dt_decreasing_rate;
                    }
                    else
                    {
                        last_decrease_step = md_info->sys.steps;
                    }
                }
            }
            else
            {
                find_max_frc<<<1, 1024>>>(md_info->atom_numbers, md_info->frc, d_max_force);
                hipMemcpy(&h_max_force, d_max_force, sizeof(float), hipMemcpyDeviceToHost);
                md_info->dt = sqrtf(dt_factor / h_max_force);
                if (md_info->sys.steps - last_decrease_step > 100)
                {
                    last_decrease_step = 0;
                }
            }
        }
        
        if (isnan(md_info->sys.h_potential) || (last_potential != 0 && (md_info->sys.h_potential - last_potential) / fabsf(last_potential) > 10))
        {
            md_info->sys.h_potential = last_potential;
            hipMemcpy(md_info->crd, md_info->last_crd, sizeof(VECTOR) * md_info->atom_numbers, hipMemcpyDeviceToDevice);
            hipMemset(md_info->vel, 0, sizeof(VECTOR) * md_info->atom_numbers);
            if (md_info->dt > 1e-8)
            {
                md_info->dt *= dt_decreasing_rate;
            }
            else
            {
                last_decrease_step = md_info->sys.steps;
            }
        }
        else
        {
            last_potential = md_info->sys.h_potential;
            hipMemcpy(md_info->last_crd, md_info->crd, sizeof(VECTOR) * md_info->atom_numbers, hipMemcpyDeviceToDevice);
        }
    }

    if (max_move <= 0)
    {
        MD_Iteration_Gradient_Descent << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
            (md_info->atom_numbers, md_info->crd, md_info->frc, md_info->d_mass_inverse, md_info->dt, md_info->vel, momentum_keep);
    }
    else
    {
        MD_Iteration_Gradient_Descent_With_Max_Move << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
            (md_info->atom_numbers, md_info->crd, md_info->frc, md_info->d_mass_inverse, md_info->dt, md_info->vel, momentum_keep, max_move);
    }
}

void MD_INFORMATION::MINIMIZATION_iteration::Check_Nan()
{
    if (dynamic_dt)
    {
        float temperature = md_info->res.Get_Residue_Temperature();
        if (isnan(temperature) || (last_temperature > 0.1 && temperature / last_temperature > 10) || (last_temperature < 0.01 && temperature > 0.1))
        {
            md_info->sys.h_potential = last_potential;
            hipMemcpy(md_info->crd, md_info->last_crd, sizeof(VECTOR) * md_info->atom_numbers, hipMemcpyDeviceToDevice);
            hipMemset(md_info->vel, 0, sizeof(VECTOR) * md_info->atom_numbers);
            if (md_info->dt > 1e-8)
            {
                md_info->dt *= dt_decreasing_rate;
            }
            else
            {
                last_decrease_step = md_info->sys.steps;
            }
        }
        last_temperature = temperature;
    }
}

void MD_INFORMATION::RERUN_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    this->md_info = md_info;
    if (md_info->mode == RERUN)
    {
        controller->printf("    Start initializing rerun:\n");
        if (!Open_File_Safely(&traj_file, controller->Command(TRAJ_COMMAND), "rb"))
        {
            controller->Throw_SPONGE_Error(spongeErrorMissingCommand,
                "MD_INFORMATION::RERUN_information::Initial",
                "Reason:\n\tno trajectory information found (command 'crd' required)");
        }
        else
        {
            controller->printf("        Open rerun coordinate trajectory\n");
        }
        controller->Set_File_Buffer(traj_file, sizeof(VECTOR) * md_info->atom_numbers);
        if (!Open_File_Safely(&box_file, controller->Command(BOX_TRAJ_COMMAND), "r"))
        {
            controller->Throw_SPONGE_Error(spongeErrorMissingCommand,
                "MD_INFORMATION::RERUN_information::Initial",
                "Reason:\n\tno box information found (command 'box' required)");
        }
        else
        {
            controller->printf("        Open rerun box trajectory\n");
        }
        controller->Set_File_Buffer(box_file, sizeof(char) * 50);
        start_frame = 0;
        if (controller->Command_Exist("rerun_start"))
        {
            controller->Check_Float("rerun_start", "MD_INFORMATION::RERUN_information::Initial");
            start_frame = atoi(controller->Command("rerun_start"));
        }
        strip_frame = 0;
        if (controller->Command_Exist("rerun_strip"))
        {
            controller->Check_Float("rerun_strip", "MD_INFORMATION::RERUN_information::Initial");
            strip_frame = atoi(controller->Command("rerun_strip"));
        }
        md_info->sys.step_limit = INT_MAX;
        controller->printf("    End initializing rerun\n\n");
    }
}

void MD_INFORMATION::RERUN_information::Iteration(int strip)
{
    int n;
    int scanf_box;
    if (strip < 0)
    {
        strip = this->strip_frame;
    }
    for (int i = 0; i < strip; i++)
    {
        n = fread(this->md_info->coordinate, sizeof(VECTOR), this->md_info->atom_numbers, traj_file);
        if (md_info->pbc.pbc)
        {
            scanf_box = fscanf(box_file, "%f %f %f %*f %*f %*f",
                &box_length_change_factor.x,
                &box_length_change_factor.y,
                &box_length_change_factor.z);
        }
        else
            box_length_change_factor = md_info->sys.box_length;
        md_info->sys.steps += 1;
    }
    n = fread(this->md_info->coordinate, sizeof(VECTOR), this->md_info->atom_numbers, traj_file);
    if (md_info->pbc.pbc)
    {
        scanf_box = fscanf(box_file, "%f %f %f %*f %*f %*f",
            &box_length_change_factor.x,
            &box_length_change_factor.y,
            &box_length_change_factor.z);
    }
    else
    {
        box_length_change_factor = md_info->sys.box_length;
        scanf_box = 3;
    }
    if (n != this->md_info->atom_numbers || scanf_box != 3)
    {
        md_info->sys.step_limit = md_info->sys.steps;
    }
    box_length_change_factor = box_length_change_factor / md_info->sys.box_length;  
    hipMemcpy(this->md_info->crd, this->md_info->coordinate, sizeof(VECTOR)* this->md_info->atom_numbers, hipMemcpyHostToDevice);
}

void MD_INFORMATION::NVE_iteration::Velocity_Verlet_1()
{
    MD_Iteration_Speed_Verlet_1 << <ceilf((float)md_info->atom_numbers / 32), 32 >> >
        (md_info->atom_numbers, 0.5*md_info->dt, md_info->dt, md_info->acc, md_info->vel, md_info->crd, md_info->frc);
}

void MD_INFORMATION::NVE_iteration::Velocity_Verlet_2()
{
    if (max_velocity <= 0)
    {
        MD_Iteration_Speed_Verlet_2 << <ceilf((float)md_info->atom_numbers / 32), 32 >> >
            (md_info->atom_numbers, 0.5*md_info->dt, md_info->d_mass_inverse, md_info->frc, md_info->vel, md_info->acc);
    }
    else
    {
        MD_Iteration_Speed_Verlet_2_With_Max_Velocity << <ceilf((float)md_info->atom_numbers / 32), 32 >> >
            (md_info->atom_numbers, 0.5*md_info->dt, md_info->d_mass_inverse, md_info->frc, md_info->vel, md_info->acc, max_velocity);
    }
}


float MD_INFORMATION::system_information::Get_Total_Atom_Ek(int is_download)
{
    MD_Atom_Ek << <ceilf((float)md_info->atom_numbers / 32.), 32 >> >(md_info->atom_numbers, md_info->d_atom_ek, md_info->vel, md_info->d_mass);
    Sum_Of_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_ek, d_sum_of_atom_ek);
    if (is_download)
    {
        hipMemcpy(&h_sum_of_atom_ek, d_sum_of_atom_ek, sizeof(float), hipMemcpyDeviceToHost);
        return h_sum_of_atom_ek;
    }
    else
    {
        return 0;
    }    
}

float MD_INFORMATION::system_information::Get_Atom_Temperature()
{
    h_temperature = Get_Total_Atom_Ek() * 2. / CONSTANT_kB / freedom;
    return h_temperature;
}

float MD_INFORMATION::residue_information::Get_Total_Residue_Ek(int is_download)
{
    MD_Residue_Ek << <ceilf((float)residue_numbers / 32.), 32 >> >
        (residue_numbers, d_res_start, d_res_end, res_ek_energy,
        md_info->vel, md_info->d_mass);
    Sum_Of_List << <1, 1024 >> >(residue_numbers, res_ek_energy, sigma_of_res_ek);
    if (is_download)
    {
        hipMemcpy(h_sigma_of_res_ek, sigma_of_res_ek, sizeof(float), hipMemcpyDeviceToHost);
        return h_sigma_of_res_ek[0];
    }
    else
    {
        return 0;
    }
}

float MD_INFORMATION::residue_information::Get_Residue_Temperature()
{
    h_temperature = Get_Total_Residue_Ek() * 2. / CONSTANT_kB / residue_numbers / 3;
    return h_temperature;
}

void MD_INFORMATION::residue_information::Residue_Crd_Map(VECTOR scaler)
{
    Get_Origin << <(residue_numbers + 1023) / 1024, 1024 >> >
        (residue_numbers, d_res_start, d_res_end,
            md_info->crd, md_info->d_mass, d_mass_inverse, d_center_of_mass);
    Map_Center_Of_Mass << < (residue_numbers + 63) / 64, { 64, 16 } >> >
        (residue_numbers, d_res_start, d_res_end, scaler, 
        d_center_of_mass, md_info->sys.box_length, md_info->crd, (int*)NULL);
}


void MD_INFORMATION::MD_Reset_Atom_Energy_And_Virial_And_Force()
{
    need_potential = 0;
    hipMemset(d_atom_energy, 0, sizeof(float)* atom_numbers);
    hipMemset(sys.d_potential, 0, sizeof(float));

    need_pressure = 0;
    hipMemset(d_atom_virial, 0, sizeof(float)* atom_numbers);
    hipMemset(sys.d_virial, 0, sizeof(float));

    hipMemset(frc, 0, sizeof(VECTOR)* atom_numbers);
}


void MD_INFORMATION::Calculate_Pressure_And_Potential_If_Needed(int is_download)
{
    if (need_pressure > 0)
    {
        sys.Get_Pressure(is_download);
    }
    if (need_potential > 0)
    {
        sys.Get_Potential(is_download);
    }
}

void MD_INFORMATION::Scale_Position_To_Center(float scaler)
{
    Scale_Position_To_Center_CUDA << <(atom_numbers + 1023) / 1024, 1024 >> > (atom_numbers, scaler, crd, 0.5f * scaler * sys.box_length);
}

float MD_INFORMATION::system_information::Get_Pressure(int is_download)
{
    //计算动能
    MD_Atom_Ek << <ceilf((float)md_info->atom_numbers / 32.), 32 >> >(md_info->atom_numbers, md_info->d_atom_ek, md_info->vel, md_info->d_mass);
    Sum_Of_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_ek, d_sum_of_atom_ek);

    //计算维里
    Add_Sum_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_virial, d_virial);

    //合并起来
    Calculate_Pressure_Cuda << <1, 1 >> >(1.0 / Get_Volume(), d_sum_of_atom_ek, d_virial, d_pressure);

    if (is_download)
    {
        hipMemcpy(&h_pressure, d_pressure, sizeof(float), hipMemcpyDeviceToHost);
        return h_pressure;
    }
    else
    {
        return 0;
    }
}

float MD_INFORMATION::system_information::Get_Potential(int is_download)
{

    Add_Sum_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_energy, d_potential);

    if (is_download)
    {
        hipMemcpy(&h_potential, d_potential, sizeof(float), hipMemcpyDeviceToHost);
        return h_potential;
    }
    else
    {
        return 0;
    }
}

void MD_INFORMATION::MD_Information_Frc_Device_To_Host()
{
    hipMemcpy(this->force, this->frc, sizeof(VECTOR)*this->atom_numbers, hipMemcpyDeviceToHost);
}

void MD_INFORMATION::MD_Information_Frc_Host_To_Device()
{
    hipMemcpy(this->frc, this->force, sizeof(VECTOR)*this->atom_numbers, hipMemcpyHostToDevice);
}

void MD_INFORMATION::Crd_Vel_Device_To_Host(int forced)
{
    if (output.current_crd_synchronized_step != sys.steps || forced)
    {    
        output.current_crd_synchronized_step = sys.steps;
        if (pbc.pbc)
            hipMemcpy(this->coordinate, this->crd, sizeof(VECTOR)*this->atom_numbers, hipMemcpyDeviceToHost);
        else
            hipMemcpy(this->coordinate, this->pbc.nopbc_crd, sizeof(VECTOR)*this->atom_numbers, hipMemcpyDeviceToHost);
        hipMemcpy(this->velocity, this->vel, sizeof(VECTOR)*this->atom_numbers, hipMemcpyDeviceToHost);
    }
}

void MD_INFORMATION::Step_Print(CONTROLLER* controller)
{
    if (mode == RERUN)
    {
        controller->Step_Print("frame", this->sys.steps);
    }
    else
    {
        controller->Step_Print("step", this->sys.steps);
        controller->Step_Print("time", this->sys.Get_Current_Time());
        controller->Step_Print("temperature", this->res.Get_Residue_Temperature());
    }

}

void MD_INFORMATION::Clear()
{

}

static void Get_Atom_Group_From_Edges(const int atom_numbers, const int* edges,
    const int* first_edge, const int* edge_next, CPP_ATOM_GROUP& mol_atoms, int* belongs)
{

    std::deque<int> queue;
    std::vector<int> visited(atom_numbers, 0);
    int atom;
    int edge_count;
    for (int i = 0; i < atom_numbers; i++)
    {
        if (!visited[i])
        {
            std::vector<int> atoms;
            visited[i] = 1;
            queue.push_back(i);
            while (!queue.empty())
            {
                atom = queue[0];
                belongs[atom] = mol_atoms.size();
                atoms.push_back(atom);
                queue.pop_front();
                edge_count = first_edge[atom];
                while (edge_count != -1)
                {
                    atom = edges[edge_count];
                    if (!visited[atom])
                    {
                        queue.push_back(atom);
                        visited[atom] = 1;
                    }
                    edge_count = edge_next[edge_count];
                }
            }
            mol_atoms.push_back(atoms);
        }
    }
}

static void Get_Molecule_Atoms(CONTROLLER* controller, int atom_numbers, CONECT connectivity,
    CPP_ATOM_GROUP& mol_atoms, std::vector<int>& molecule_belongings)
{
    //分子拓扑是一个无向图，邻接表进行描述
    int edge_numbers = 0;
    for (int i = 0; i < atom_numbers; i++)
    {
        edge_numbers += connectivity[i].size();
    }
    edge_numbers *= 2;
    int* first_edge = NULL; //每个原子的第一个边（链表的头）
    int* edges = NULL;  //每个边的序号
    int* edge_next = NULL; //每个原子的边（链表结构）
    Malloc_Safely((void**)&first_edge, sizeof(int) * atom_numbers);
    Malloc_Safely((void**)&edges, sizeof(int) * edge_numbers);
    Malloc_Safely((void**)&edge_next, sizeof(int) * edge_numbers);
    //初始化链表
    for (int i = 0; i < atom_numbers; i++)
    {
        first_edge[i] = -1;
    }
    int atom_i, atom_j, edge_count = 0;
    for (int atom_i = 0; atom_i < atom_numbers; atom_i++)
    {
        std::set<int> conect_i = connectivity[atom_i];
        for (auto iter = conect_i.begin(); iter != conect_i.end(); iter++)
        {
            atom_j = *iter;
            edge_next[edge_count] = first_edge[atom_i];
            first_edge[atom_i] = edge_count;
            edges[edge_count] = atom_j;
            edge_count++;
        }
    }
    if (controller->Command_Exist("make_output_whole"))
    {
        std::string temp = string_strip(controller->Original_Command("make_output_whole"));
        for (std::string aword : string_split(temp, " "))
        {
            std::vector<std::string> atomij = string_split(string_strip(aword), "-");
            if (atomij.size() != 2 || !is_str_int(atomij[0].c_str()) || !is_str_int(atomij[1].c_str()))
            {
                controller->Throw_SPONGE_Error(spongeErrorValueErrorCommand, "Move_Crd_Nearest_From_Exclusions_Host",
                    "Reason:\n\t'make_output_whole' should provide atoms in the format of atom_i-atom_j");
            }
            atom_i = atoi(atomij[0].c_str());
            atom_j = atoi(atomij[1].c_str());
            edge_next[edge_count] = first_edge[atom_i];
            first_edge[atom_i] = edge_count;
            edges[edge_count] = atom_j;
            edge_count++;
            edge_next[edge_count] = first_edge[atom_j];
            first_edge[atom_j] = edge_count;
            edges[edge_count] = atom_i;
            edge_count++;
        }
    }
    Get_Atom_Group_From_Edges(atom_numbers, edges, first_edge, edge_next, mol_atoms, &molecule_belongings[0]);
    free(first_edge);
    free(edges);
    free(edge_next);
}

static std::vector<int> Check_Periodic_Molecules(CPP_ATOM_GROUP mol_atoms, const VECTOR* crd, const VECTOR box_length)
{
    std::vector<int> periodic_mols;
    std::vector<int> atoms;
    VECTOR abosolute_dr;
    for (int i = 0; i < mol_atoms.size(); i++)
    {
        atoms = mol_atoms[i];
        abosolute_dr.x = 0;
        abosolute_dr.y = 0;
        abosolute_dr.z = 0;
        for (int j = 1; j < atoms.size(); j++)
        {
            int atom = atoms[j];
            int atom_front = atoms[j - 1];
            abosolute_dr = abosolute_dr + Get_Periodic_Displacement(crd[atom], crd[atom_front], box_length);
        }
        periodic_mols.push_back(abosolute_dr.x * abosolute_dr.x >= box_length.x * box_length.x
            || abosolute_dr.y * abosolute_dr.y >= box_length.y * box_length.y
            || abosolute_dr.z * abosolute_dr.z >= box_length.z * box_length.z);
    }
    return periodic_mols;
}

static void Move_Crd_Nearest_From_Connectivity(CPP_ATOM_GROUP mol_atoms, VECTOR* crd, 
    const VECTOR box_length, std::vector<int> periodic_molecules)
{
    std::vector<int> atoms;
    for (int i = 0; i < mol_atoms.size(); i++)
    {
        if (periodic_molecules[i])
        {
            continue;
        }
        atoms = mol_atoms[i];
        for (int j = 1; j < atoms.size(); j++)
        {
            int atom = atoms[j];
            int atom_front = atoms[j - 1];
            int map_x = floorf((crd[atom_front].x - crd[atom].x) / box_length.x + 0.5);
            crd[atom].x = crd[atom].x + map_x * box_length.x;
            map_x = floorf((crd[atom_front].y - crd[atom].y) / box_length.y + 0.5);
            crd[atom].y = crd[atom].y + map_x * box_length.y;
            map_x = floorf((crd[atom_front].z - crd[atom].z) / box_length.z + 0.5);
            crd[atom].z = crd[atom].z + map_x * box_length.z;
        }
    }
}

void MD_INFORMATION::molecule_information::Initial(CONTROLLER *controller)
{
    if (!md_info->pbc.pbc)
        return;
    //分子拓扑是一个无向图，邻接表进行描述，通过排除表形成
    CPP_ATOM_GROUP mol_atoms;
    std::vector<int> molecule_belongings(md_info->atom_numbers, 0);
    Get_Molecule_Atoms(controller, md_info->atom_numbers, md_info->sys.connectivity,
        mol_atoms, molecule_belongings);
    molecule_numbers = mol_atoms.size();
    std::vector<int> h_periodicity = Check_Periodic_Molecules(mol_atoms, md_info->coordinate, md_info->sys.box_length);
    Cuda_Malloc_And_Copy_Safely((void**)&d_periodicity, &h_periodicity[0],
        sizeof(int) * molecule_numbers);
    Move_Crd_Nearest_From_Connectivity(mol_atoms, md_info->coordinate, md_info->sys.box_length, h_periodicity);
    hipMemcpy(md_info->crd, md_info->coordinate, sizeof(VECTOR) * md_info->atom_numbers, hipMemcpyHostToDevice);

    Malloc_Safely((void**)&h_mass, sizeof(float)*molecule_numbers);
    Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*molecule_numbers);
    Malloc_Safely((void**)&h_atom_start, sizeof(int)*molecule_numbers);
    Malloc_Safely((void**)&h_atom_end, sizeof(int)*molecule_numbers);
    Malloc_Safely((void**)&h_residue_start, sizeof(int)*molecule_numbers);
    Malloc_Safely((void**)&h_residue_end, sizeof(int)*molecule_numbers);
    Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*molecule_numbers);

    Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*molecule_numbers);
    Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*molecule_numbers);
    Cuda_Malloc_Safely((void**)&d_atom_start, sizeof(int)*molecule_numbers);
    Cuda_Malloc_Safely((void**)&d_atom_end, sizeof(int)*molecule_numbers);
    Cuda_Malloc_Safely((void**)&d_residue_start, sizeof(int)*molecule_numbers);
    Cuda_Malloc_Safely((void**)&d_residue_end, sizeof(int)*molecule_numbers);
    Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*molecule_numbers);

    int molecule_j = 0;
    h_atom_start[0] = 0;
    //该判断基于一个分子的所有原子一定在列表里是连续的
    for (int i = 0; i < md_info->atom_numbers; i++)
    {
        if (molecule_belongings[i] != molecule_j)
        {
            if (molecule_belongings[i] < molecule_j)
            {
                char error_reason[CHAR_LENGTH_MAX];
                sprintf(error_reason, "Reason:\n\tthe indexes of atoms in the same one molecule should be continous, and atom #%d is not right\n", i);
                controller->Throw_SPONGE_Error(spongeErrorValueErrorCommand, "MD_INFORMATION::molecule_information::Initial", error_reason);
            }
            h_atom_end[molecule_j] = i;
            molecule_j += 1;
            if (molecule_j < molecule_numbers)
                h_atom_start[molecule_j] = i;
        }
    }
    h_atom_end[molecule_numbers - 1] = md_info->atom_numbers;
    
    molecule_j = 0;
    h_residue_start[0] = 0;
    //该判断基于一个分子的所有残基一定在列表里是连续的，且原子在残基里也是连续的
    for (int i = 0; i < md_info->res.residue_numbers; i++)
    {
        if (md_info->res.h_res_start[i] == h_atom_end[molecule_j])
        {
            h_residue_end[molecule_j] = i;
            molecule_j += 1;
            if (molecule_j < molecule_numbers)
                h_residue_start[molecule_j] = i;
        }
    }
    h_residue_end[molecule_numbers - 1] = md_info->res.residue_numbers;

    for (int i = 0; i < molecule_numbers; i++)
    {
        h_mass[i] = 0;
        for (molecule_j = h_atom_start[i]; molecule_j < h_atom_end[i]; molecule_j++)
        {
            h_mass[i] += md_info->h_mass[molecule_j];
        }
        h_mass_inverse[i] = 1.0f / h_mass[i];
    }

    hipMemcpy(d_mass, h_mass, sizeof(float)*molecule_numbers, hipMemcpyHostToDevice);
    hipMemcpy(d_mass_inverse, h_mass_inverse, sizeof(float)*molecule_numbers, hipMemcpyHostToDevice);
    hipMemcpy(d_atom_start, h_atom_start, sizeof(int)*molecule_numbers, hipMemcpyHostToDevice);
    hipMemcpy(d_atom_end, h_atom_end, sizeof(int)*molecule_numbers, hipMemcpyHostToDevice);
    hipMemcpy(d_residue_start, h_residue_start, sizeof(int)*molecule_numbers, hipMemcpyHostToDevice);
    hipMemcpy(d_residue_end, h_residue_end, sizeof(int)*molecule_numbers, hipMemcpyHostToDevice);
}

static __global__ void NoPBC_Mapping(const int atom_numbers, VECTOR* crd, VECTOR* last_crd,
 VECTOR* nopbc_crd, VECTOR box_length)
{
    int atom_i = threadIdx.x + blockDim.x * blockIdx.x;
    if (atom_i < atom_numbers)
    {
        VECTOR crd_i = crd[atom_i];
        nopbc_crd[atom_i] = nopbc_crd[atom_i] + Get_Periodic_Displacement(crd_i, last_crd[atom_i], box_length);
        crd_i.x = crd_i.x - floorf(crd_i.x / box_length.x) * box_length.x;
        crd_i.y = crd_i.y - floorf(crd_i.y / box_length.y) * box_length.y;
        crd_i.z = crd_i.z - floorf(crd_i.z / box_length.z) * box_length.z;         
        last_crd[atom_i] = crd_i;
        crd[atom_i] = crd_i;
    }
}

void MD_INFORMATION::molecule_information::Molecule_Crd_Map(float scaler)
{
    if (!md_info->pbc.pbc)
    {
        NoPBC_Mapping<< <(md_info->atom_numbers + 1023) / 1024, 1024 >> >(md_info->atom_numbers,
            md_info->crd, md_info->last_crd, md_info->pbc.nopbc_crd, md_info->sys.box_length); 
    }
    else
    {
        //为了有一个分子有很多残基，而其他分子都很小这种情况的并行，先求残基的质心
        Get_Origin << <(md_info->res.residue_numbers + 1023) / 1024, 1024 >> >
            (md_info->res.residue_numbers, md_info->res.d_res_start, md_info->res.d_res_end, 
                md_info->crd, md_info->d_mass, md_info->res.d_mass_inverse, md_info->res.d_center_of_mass);
        //再用残基的质心求分子的质心
        Get_Origin << <(molecule_numbers + 1023) / 1024, 1024 >> >
            (molecule_numbers, d_residue_start, d_residue_end, md_info->res.d_center_of_mass,
                md_info->res.d_mass, d_mass_inverse, d_center_of_mass);

        Map_Center_Of_Mass << < (molecule_numbers + 63) / 64, { 64, 16 } >> > (molecule_numbers, d_atom_start, d_atom_end,
            scaler, d_center_of_mass, md_info->sys.box_length, md_info->crd, d_periodicity);
    }
}


void MD_INFORMATION::molecule_information::Molecule_Crd_Map(VECTOR scaler)
{
    if (!md_info->pbc.pbc)
    {
        NoPBC_Mapping<< <(md_info->atom_numbers + 1023) / 1024, 1024 >> >(md_info->atom_numbers,
            md_info->crd, md_info->last_crd, md_info->pbc.nopbc_crd, md_info->sys.box_length); 
    }
    else
    {
        //为了有一个分子有很多残基，而其他分子都很小这种情况的并行，先求残基的质心
        Get_Origin << <(md_info->res.residue_numbers + 1023) / 1024, 1024 >> >
            (md_info->res.residue_numbers, md_info->res.d_res_start, md_info->res.d_res_end,
                md_info->crd, md_info->d_mass, md_info->res.d_mass_inverse, md_info->res.d_center_of_mass);
        //再用残基的质心求分子的质心
        Get_Origin << <(molecule_numbers + 1023) / 1024, 1024 >> >
            (molecule_numbers, d_residue_start, d_residue_end, md_info->res.d_center_of_mass,
                md_info->res.d_mass, d_mass_inverse, d_center_of_mass);
        Map_Center_Of_Mass << < (molecule_numbers + 63) / 64, { 64, 16 } >> > (molecule_numbers, d_atom_start, d_atom_end, 
            scaler, d_center_of_mass, md_info->sys.box_length, md_info->crd, d_periodicity);
    }
}
