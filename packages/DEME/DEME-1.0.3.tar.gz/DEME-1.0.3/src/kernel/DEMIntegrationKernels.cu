#include "hip/hip_runtime.h"
// DEM integration related custom kernels
#include <DEMHelperKernels.cu>
#include <DEM/Defines.h>

// Apply presecibed velocity and report whether the `true' physics should be skipped, rather than added on top of that
template <typename T1, typename T2>
inline __device__ void applyPrescribedVel(bool& LinXPrescribed,
                                          bool& LinYPrescribed,
                                          bool& LinZPrescribed,
                                          bool& RotXPrescribed,
                                          bool& RotYPrescribed,
                                          bool& RotZPrescribed,
                                          T1& vX,
                                          T1& vY,
                                          T1& vZ,
                                          T2& omgBarX,
                                          T2& omgBarY,
                                          T2& omgBarZ,
                                          const deme::family_t& family,
                                          const float& t) {
    switch (family) {
        _velPrescriptionStrategy_;
        default:
            // Default can just do nothing
            return;
    }
}

// Apply presecibed location and report whether the `true' physics should be skipped, rather than added on top of that
template <typename T1, typename T2>
inline __device__ void applyPrescribedPos(bool& LinXPrescribed,
                                          bool& LinYPrescribed,
                                          bool& LinZPrescribed,
                                          bool& RotPrescribed,
                                          T1& X,
                                          T1& Y,
                                          T1& Z,
                                          T2& oriQw,
                                          T2& oriQx,
                                          T2& oriQy,
                                          T2& oriQz,
                                          const deme::family_t& family,
                                          const float& t) {
    switch (family) {
        _posPrescriptionStrategy_;
        default:
            // Default can just do nothing
            return;
    }
}

// Apply extra accelerations for family numbers
template <typename T1, typename T2>
inline __device__ void applyAddedAcceleration(T1& accX,
                                              T1& accY,
                                              T1& accZ,
                                              T2& angAccX,
                                              T2& angAccY,
                                              T2& angAccZ,
                                              const deme::family_t& family,
                                              const float& t) {
    switch (family) {
        _accPrescriptionStrategy_;
        default:
            return;
    }
}

inline __device__ void integrateVel(deme::bodyID_t thisClump,
                                    deme::DEMSimParams* simParams,
                                    deme::DEMDataDT* granData,
                                    float3& v,
                                    float3& omgBar,
                                    float h,
                                    float t) {
    deme::family_t family_code = granData->familyID[thisClump];
    bool LinXPrescribed = false, LinYPrescribed = false, LinZPrescribed = false, RotXPrescribed = false,
         RotYPrescribed = false, RotZPrescribed = false;

    // Keep tab of the old... we'll need that
    float3 old_v = make_float3(granData->vX[thisClump], granData->vY[thisClump], granData->vZ[thisClump]);
    float3 old_omgBar =
        make_float3(granData->omgBarX[thisClump], granData->omgBarY[thisClump], granData->omgBarZ[thisClump]);

    // The user may directly change v and omgBar info in global memory in applyPrescribedVel
    applyPrescribedVel<float, float>(LinXPrescribed, LinYPrescribed, LinZPrescribed, RotXPrescribed, RotYPrescribed,
                                     RotZPrescribed, granData->vX[thisClump], granData->vY[thisClump],
                                     granData->vZ[thisClump], granData->omgBarX[thisClump],
                                     granData->omgBarY[thisClump], granData->omgBarZ[thisClump], family_code, (float)t);

    float3 v_update = make_float3(0, 0, 0), omgBar_update = make_float3(0, 0, 0);
    float3 extra_acc = make_float3(0, 0, 0), extra_angAcc = make_float3(0, 0, 0);
    // User's addition of accelerations won't affect acc arrays in global memory; that is, if the user query the contact
    // acceleration, still they don't get the part they applied in this acc prescription
    applyAddedAcceleration<float, float>(extra_acc.x, extra_acc.y, extra_acc.z, extra_angAcc.x, extra_angAcc.y,
                                         extra_angAcc.z, family_code, (float)t);

    if (!LinXPrescribed) {
        v_update.x = (granData->aX[thisClump] + extra_acc.x + simParams->Gx) * h;
        granData->vX[thisClump] += v_update.x;
    } else {
        old_v.x = granData->vX[thisClump];
    }
    if (!LinYPrescribed) {
        v_update.y = (granData->aY[thisClump] + extra_acc.y + simParams->Gy) * h;
        granData->vY[thisClump] += v_update.y;
    } else {
        old_v.y = granData->vY[thisClump];
    }
    if (!LinZPrescribed) {
        v_update.z = (granData->aZ[thisClump] + extra_acc.z + simParams->Gz) * h;
        granData->vZ[thisClump] += v_update.z;
    } else {
        old_v.z = granData->vZ[thisClump];
    }

    if (!RotXPrescribed) {
        omgBar_update.x = (granData->alphaX[thisClump] + extra_angAcc.x) * h;
        granData->omgBarX[thisClump] += omgBar_update.x;
    } else {
        old_omgBar.x = granData->omgBarX[thisClump];
    }
    if (!RotYPrescribed) {
        omgBar_update.y = (granData->alphaY[thisClump] + extra_angAcc.y) * h;
        granData->omgBarY[thisClump] += omgBar_update.y;
    } else {
        old_omgBar.y = granData->omgBarY[thisClump];
    }
    if (!RotZPrescribed) {
        omgBar_update.z = (granData->alphaZ[thisClump] + extra_angAcc.z) * h;
        granData->omgBarZ[thisClump] += omgBar_update.z;
    } else {
        old_omgBar.z = granData->omgBarZ[thisClump];
    }

    // We need to set v and omgBar, and they will be used in position/quaternion update
    _integrationVelocityPassOnStrategy_;
}

// inline __device__ void locateNewVoxel(deme::voxelID_t& voxel, int64_t& locX_tmp, int64_t& locY_tmp, int64_t&
// locZ_tmp) {
//     deme::voxelID_t voxelX;
//     deme::voxelID_t voxelY;
//     deme::voxelID_t voxelZ;
//     IDChopper<deme::voxelID_t, deme::voxelID_t>(voxelX, voxelY, voxelZ, voxel, _nvXp2_, _nvYp2_);

//     // MAX_SUBVOXEL is int64 and large enough to handle VOXEL_RES_POWER2 == 16 or 32
//     voxelX += div_floor<int64_t, int64_t>(locX_tmp, deme::MAX_SUBVOXEL);
//     voxelY += div_floor<int64_t, int64_t>(locY_tmp, deme::MAX_SUBVOXEL);
//     voxelZ += div_floor<int64_t, int64_t>(locZ_tmp, deme::MAX_SUBVOXEL);
//     locX_tmp = mod_floor<int64_t, int64_t>(locX_tmp, deme::MAX_SUBVOXEL);
//     locY_tmp = mod_floor<int64_t, int64_t>(locY_tmp, deme::MAX_SUBVOXEL);
//     locZ_tmp = mod_floor<int64_t, int64_t>(locZ_tmp, deme::MAX_SUBVOXEL);

//     IDPacker<deme::voxelID_t, deme::voxelID_t>(voxel, voxelX, voxelY, voxelZ, _nvXp2_, _nvYp2_);
// }

inline __device__ void integratePos(deme::bodyID_t thisClump,
                                    deme::DEMDataDT* granData,
                                    float3 v,
                                    float3 omgBar,
                                    float h,
                                    float t) {
    // This block is not needed, with our current way of integration...
    // int64_t locX_tmp = (int64_t)granData->locX[thisClump];
    // int64_t locY_tmp = (int64_t)granData->locY[thisClump];
    // int64_t locZ_tmp = (int64_t)granData->locZ[thisClump];
    // locateNewVoxel(newVoxel, locX_tmp, locY_tmp, locZ_tmp);
    // locX_tmp += (int64_t)((double)v.x / _l_ * h);
    // locY_tmp += (int64_t)((double)v.y / _l_ * h);
    // locZ_tmp += (int64_t)((double)v.z / _l_ * h);

    double X, Y, Z;
    // Now XYZ gets the old position. We can write them directly back, then it is equivalent to being LinPrescribed.
    voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
        X, Y, Z, granData->voxelID[thisClump], granData->locX[thisClump], granData->locY[thisClump],
        granData->locZ[thisClump], _nvXp2_, _nvYp2_, _voxelSize_, _l_);

    deme::family_t family_code = granData->familyID[thisClump];
    bool LinXPrescribed = false, LinYPrescribed = false, LinZPrescribed = false, RotPrescribed = false;
    applyPrescribedPos<double, deme::oriQ_t>(
        LinXPrescribed, LinYPrescribed, LinZPrescribed, RotPrescribed, X, Y, Z, granData->oriQw[thisClump],
        granData->oriQx[thisClump], granData->oriQy[thisClump], granData->oriQz[thisClump], family_code, (float)t);

    if (!LinXPrescribed) {
        // Impllicitly, pos integration strategy is here
        X += (double)v.x * h;
    }
    if (!LinYPrescribed) {
        Y += (double)v.y * h;
    }
    if (!LinZPrescribed) {
        Z += (double)v.z * h;
    }
    positionToVoxelID<deme::voxelID_t, deme::subVoxelPos_t, double>(
        granData->voxelID[thisClump], granData->locX[thisClump], granData->locY[thisClump], granData->locZ[thisClump],
        X, Y, Z, _nvXp2_, _nvYp2_, _voxelSize_, _l_);

    if (!RotPrescribed) {
        // Then integrate the quaternion
        // 1st Taylor series multiplier. First use it to record delta rotation...
        // Refer to https://stackoverflow.com/questions/24197182/efficient-quaternion-angular-velocity/24201879#24201879
        const float3 ha = 0.5 * h * omgBar;
        float4 oriQ = make_float4(ha.x, ha.y, ha.z, 1.0);  // xyzw
        // Note: Yes it is Quat * deltaRot, not the other way around. Then store result in oriQ.
        HamiltonProduct(oriQ.w, oriQ.x, oriQ.y, oriQ.z, granData->oriQw[thisClump], granData->oriQx[thisClump],
                        granData->oriQy[thisClump], granData->oriQz[thisClump], oriQ.w, oriQ.x, oriQ.y, oriQ.z);
        // Normalizing it is essential. Note even if you use an exp map to update quaternion, you still need to
        // normalize.
        oriQ /= length(oriQ);
        granData->oriQw[thisClump] = oriQ.w;
        granData->oriQx[thisClump] = oriQ.x;
        granData->oriQy[thisClump] = oriQ.y;
        granData->oriQz[thisClump] = oriQ.z;
    }
}

__global__ void integrateOwners(deme::DEMSimParams* simParams, deme::DEMDataDT* granData) {
    deme::bodyID_t thisClump = blockIdx.x * blockDim.x + threadIdx.x;
    if (thisClump < simParams->nOwnerBodies) {
        // These 2 quantities mean the velocity and ang vel used for updating position/quaternion for this step.
        // Depending on the integration scheme in use, they can be different.
        float3 v, omgBar;
        integrateVel(thisClump, simParams, granData, v, omgBar, (float)simParams->h, (float)simParams->timeElapsed);
        integratePos(thisClump, granData, v, omgBar, (float)simParams->h, (float)simParams->timeElapsed);
    }
}
