#include "hip/hip_runtime.h"
// DEM integration related custom kernels
#include <DEMHelperKernels.cu>
#include <DEM/Defines.h>
_kernelIncludes_

// Apply presecibed velocity and report whether the `true' physics should be skipped, rather than added on top of
// that
template <typename T1, typename T2, typename T3, typename T4>
inline __device__ void applyPrescribedVel(bool& LinVelXPrescribed,
                                          bool& LinVelYPrescribed,
                                          bool& LinVelZPrescribed,
                                          bool& RotVelXPrescribed,
                                          bool& RotVelYPrescribed,
                                          bool& RotVelZPrescribed,
                                          T1& vX,
                                          T1& vY,
                                          T1& vZ,
                                          T2& omgBarX,
                                          T2& omgBarY,
                                          T2& omgBarZ,
                                          T3 X,
                                          T3 Y,
                                          T3 Z,
                                          T4 oriQw,
                                          T4 oriQx,
                                          T4 oriQy,
                                          T4 oriQz,
                                          const deme::family_t& family,
                                          const float& t) {
    switch (family) {
        _velPrescriptionStrategy_;
        default:
            // Default can just do nothing
            return;
    }
}

// Apply presecibed location and report whether the `true' physics should be skipped, rather than added on top of that
template <typename T1, typename T2, typename T3, typename T4>
inline __device__ void applyPrescribedPos(bool& LinXPrescribed,
                                          bool& LinYPrescribed,
                                          bool& LinZPrescribed,
                                          bool& RotPrescribed,
                                          T1& X,
                                          T1& Y,
                                          T1& Z,
                                          T2& oriQw,
                                          T2& oriQx,
                                          T2& oriQy,
                                          T2& oriQz,
                                          T3 vX,
                                          T3 vY,
                                          T3 vZ,
                                          T4 omgBarX,
                                          T4 omgBarY,
                                          T4 omgBarZ,
                                          const deme::family_t& family,
                                          const float& t) {
    switch (family) {
        _posPrescriptionStrategy_;
        default:
            // Default can just do nothing
            return;
    }
}

// Apply extra accelerations for family numbers
template <typename T1, typename T2, typename T3, typename T4, typename T5, typename T6>
inline __device__ void applyAddedAcceleration(T1& accX,
                                              T1& accY,
                                              T1& accZ,
                                              T2& angAccX,
                                              T2& angAccY,
                                              T2& angAccZ,
                                              T3 X,
                                              T3 Y,
                                              T3 Z,
                                              T4 oriQw,
                                              T4 oriQx,
                                              T4 oriQy,
                                              T4 oriQz,
                                              T5 vX,
                                              T5 vY,
                                              T5 vZ,
                                              T6 omgBarX,
                                              T6 omgBarY,
                                              T6 omgBarZ,
                                              const deme::family_t& family,
                                              const float& t) {
    switch (family) {
        _accPrescriptionStrategy_;
        default:
            return;
    }
}

inline __device__ void integrateVelPos(deme::bodyID_t thisClump,
                                       deme::DEMSimParams* simParams,
                                       deme::DEMDataDT* granData,
                                       float3& v,
                                       float3& omgBar,
                                       float h,
                                       float t) {
    // Acquisition phase...
    deme::family_t family_code = granData->familyID[thisClump];
    bool LinVelXPrescribed = false, LinVelYPrescribed = false, LinVelZPrescribed = false, RotVelXPrescribed = false,
         RotVelYPrescribed = false, RotVelZPrescribed = false;
    bool LinXPrescribed = false, LinYPrescribed = false, LinZPrescribed = false, RotPrescribed = false;
    double X, Y, Z;
    // Keep tab of the old... we'll need that
    float3 old_v = make_float3(granData->vX[thisClump], granData->vY[thisClump], granData->vZ[thisClump]);
    float3 old_omgBar =
        make_float3(granData->omgBarX[thisClump], granData->omgBarY[thisClump], granData->omgBarZ[thisClump]);

    {
        // Now XYZ gets the old position. We can write them directly back, then it is equivalent to being LinPrescribed.
        voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
            X, Y, Z, granData->voxelID[thisClump], granData->locX[thisClump], granData->locY[thisClump],
            granData->locZ[thisClump], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
        // Do this and we get the `true' pos... Needed for prescription
        X += (double)simParams->LBFX;
        Y += (double)simParams->LBFY;
        Z += (double)simParams->LBFZ;

        // The user may directly change v and omgBar info in global memory in applyPrescribedVel (XYZ and oriQ in this
        // call are read-only)
        applyPrescribedVel(LinVelXPrescribed, LinVelYPrescribed, LinVelZPrescribed, RotVelXPrescribed,
                           RotVelYPrescribed, RotVelZPrescribed, granData->vX[thisClump], granData->vY[thisClump],
                           granData->vZ[thisClump], granData->omgBarX[thisClump], granData->omgBarY[thisClump],
                           granData->omgBarZ[thisClump], X, Y, Z, granData->oriQw[thisClump],
                           granData->oriQx[thisClump], granData->oriQy[thisClump], granData->oriQz[thisClump],
                           family_code, (float)t);
        // The user may directly change oriQ info (vX and omgBar in this call are read-only)
        applyPrescribedPos(LinXPrescribed, LinYPrescribed, LinZPrescribed, RotPrescribed, X, Y, Z,
                           granData->oriQw[thisClump], granData->oriQx[thisClump], granData->oriQy[thisClump],
                           granData->oriQz[thisClump], granData->vX[thisClump], granData->vY[thisClump],
                           granData->vZ[thisClump], granData->omgBarX[thisClump], granData->omgBarY[thisClump],
                           granData->omgBarZ[thisClump], family_code, (float)t);
    }

    // Operation phase...

    {
        // User's addition of accelerations won't affect acc arrays in global memory; that is, if the user query the
        // contact acceleration, still they don't get the part they applied in this acc prescription
        float3 v_update = make_float3(0, 0, 0), omgBar_update = make_float3(0, 0, 0);
        float3 extra_acc = make_float3(0, 0, 0), extra_angAcc = make_float3(0, 0, 0);
        applyAddedAcceleration(extra_acc.x, extra_acc.y, extra_acc.z, extra_angAcc.x, extra_angAcc.y, extra_angAcc.z, X,
                               Y, Z, granData->oriQw[thisClump], granData->oriQx[thisClump], granData->oriQy[thisClump],
                               granData->oriQz[thisClump], granData->vX[thisClump], granData->vY[thisClump],
                               granData->vZ[thisClump], granData->omgBarX[thisClump], granData->omgBarY[thisClump],
                               granData->omgBarZ[thisClump], family_code, (float)t);

        if (!LinVelXPrescribed) {
            v_update.x = (granData->aX[thisClump] + extra_acc.x + simParams->Gx) * h;
            granData->vX[thisClump] += v_update.x;
        } else {
            old_v.x = granData->vX[thisClump];
        }
        if (!LinVelYPrescribed) {
            v_update.y = (granData->aY[thisClump] + extra_acc.y + simParams->Gy) * h;
            granData->vY[thisClump] += v_update.y;
        } else {
            old_v.y = granData->vY[thisClump];
        }
        if (!LinVelZPrescribed) {
            v_update.z = (granData->aZ[thisClump] + extra_acc.z + simParams->Gz) * h;
            granData->vZ[thisClump] += v_update.z;
        } else {
            old_v.z = granData->vZ[thisClump];
        }

        if (!RotVelXPrescribed) {
            omgBar_update.x = (granData->alphaX[thisClump] + extra_angAcc.x) * h;
            granData->omgBarX[thisClump] += omgBar_update.x;
        } else {
            old_omgBar.x = granData->omgBarX[thisClump];
        }
        if (!RotVelYPrescribed) {
            omgBar_update.y = (granData->alphaY[thisClump] + extra_angAcc.y) * h;
            granData->omgBarY[thisClump] += omgBar_update.y;
        } else {
            old_omgBar.y = granData->omgBarY[thisClump];
        }
        if (!RotVelZPrescribed) {
            omgBar_update.z = (granData->alphaZ[thisClump] + extra_angAcc.z) * h;
            granData->omgBarZ[thisClump] += omgBar_update.z;
        } else {
            old_omgBar.z = granData->omgBarZ[thisClump];
        }

        // We need to set v and omgBar, and they will be used in position/quaternion update
        _integrationVelocityPassOnStrategy_;
    }

    // With v and omgBar. update pos now...
    {
        if (!LinXPrescribed) {
            // Impllicitly, pos integration strategy is here
            X += (double)v.x * h;
        }
        if (!LinYPrescribed) {
            Y += (double)v.y * h;
        }
        if (!LinZPrescribed) {
            Z += (double)v.z * h;
        }
        // Undo the influence of LBF...
        X -= (double)simParams->LBFX;
        Y -= (double)simParams->LBFY;
        Z -= (double)simParams->LBFZ;
        positionToVoxelID<deme::voxelID_t, deme::subVoxelPos_t, double>(
            granData->voxelID[thisClump], granData->locX[thisClump], granData->locY[thisClump],
            granData->locZ[thisClump], X, Y, Z, _nvXp2_, _nvYp2_, _voxelSize_, _l_);

        if (!RotPrescribed) {
            // Then integrate the quaternion
            // 1st Taylor series multiplier. First use it to record delta rotation...
            // Refer to
            // https://stackoverflow.com/questions/24197182/efficient-quaternion-angular-velocity/24201879#24201879
            const float3 ha = 0.5 * h * omgBar;
            float4 oriQ = make_float4(ha.x, ha.y, ha.z, 1.0);  // xyzw
            // Note: Yes it is Quat * deltaRot, not the other way around. Then store result in oriQ.
            HamiltonProduct(oriQ.w, oriQ.x, oriQ.y, oriQ.z, granData->oriQw[thisClump], granData->oriQx[thisClump],
                            granData->oriQy[thisClump], granData->oriQz[thisClump], oriQ.w, oriQ.x, oriQ.y, oriQ.z);
            // Normalizing it is essential. Note even if you use an exp map to update quaternion, you still need to
            // normalize.
            oriQ /= length(oriQ);
            granData->oriQw[thisClump] = oriQ.w;
            granData->oriQx[thisClump] = oriQ.x;
            granData->oriQy[thisClump] = oriQ.y;
            granData->oriQz[thisClump] = oriQ.z;
        }
    }
}

// inline __device__ void locateNewVoxel(deme::voxelID_t& voxel, int64_t& locX_tmp, int64_t& locY_tmp, int64_t&
// locZ_tmp) {
//     deme::voxelID_t voxelX;
//     deme::voxelID_t voxelY;
//     deme::voxelID_t voxelZ;
//     IDChopper<deme::voxelID_t, deme::voxelID_t>(voxelX, voxelY, voxelZ, voxel, _nvXp2_, _nvYp2_);

//     // MAX_SUBVOXEL is int64 and large enough to handle VOXEL_RES_POWER2 == 16 or 32
//     voxelX += div_floor<int64_t, int64_t>(locX_tmp, deme::MAX_SUBVOXEL);
//     voxelY += div_floor<int64_t, int64_t>(locY_tmp, deme::MAX_SUBVOXEL);
//     voxelZ += div_floor<int64_t, int64_t>(locZ_tmp, deme::MAX_SUBVOXEL);
//     locX_tmp = mod_floor<int64_t, int64_t>(locX_tmp, deme::MAX_SUBVOXEL);
//     locY_tmp = mod_floor<int64_t, int64_t>(locY_tmp, deme::MAX_SUBVOXEL);
//     locZ_tmp = mod_floor<int64_t, int64_t>(locZ_tmp, deme::MAX_SUBVOXEL);

//     IDPacker<deme::voxelID_t, deme::voxelID_t>(voxel, voxelX, voxelY, voxelZ, _nvXp2_, _nvYp2_);
// }

__global__ void integrateOwners(deme::DEMSimParams* simParams, deme::DEMDataDT* granData) {
    deme::bodyID_t thisClump = blockIdx.x * blockDim.x + threadIdx.x;
    if (thisClump < simParams->nOwnerBodies) {
        // These 2 quantities mean the velocity and ang vel used for updating position/quaternion for this step.
        // Depending on the integration scheme in use, they can be different.
        float3 v, omgBar;
        integrateVelPos(thisClump, simParams, granData, v, omgBar, (float)simParams->h, (float)simParams->timeElapsed);
    }
}
